
#include <hip/hip_runtime.h>
#ifdef GPU_ENABLED
#include "GridGPUFluidKernel.h"
#include "GPUHelper.h"
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/pair.h>
#include <thrust/execution_policy.h>

const int kgrid_BLOCKSIZE_1D = 512;
const int kgrid_BLOCKSIZE_REDUCED = 256;
const int kgrid_NUM_NEIGHBORS = 5;
const int kgrid_MAX_CELL_SIZE = 20;

const scalar kgrid_RELAXATION = 0.01;

bool grid_deviceHappy = true;

// fluid characteristics
__constant__ scalar c_h;

// grid bounds
__constant__ scalar c_minX;
__constant__ scalar c_maxX;
__constant__ scalar c_minY;
__constant__ scalar c_maxY;
__constant__ scalar c_minZ;
__constant__ scalar c_maxZ;
__constant__ scalar c_eps;

// grid size
__constant__ int c_gridSizeX;
__constant__ int c_gridSizeY;
__constant__ int c_gridSizeZ;

///////////////////////////////////////////////
/// Function Headers
///////////////////////////////////////////////

// kernel functions
__device__ scalar kgrid_Poly6Kernel(Vector3s &pi, Vector3s &pj, scalar h);
__device__ Vector3s kgrid_SpikyKernelGrad(Vector3s &pi, Vector3s &pj, scalar h);

// gradient functions
__device__ Vector3s kgrid_calcGradConstraint(Vector3s& pi, Vector3s& pj, scalar p0, scalar h);
__device__ Vector3s kgrid_calcGradConstraintAtI(Vector3s &pi,
                                                Vector3s* neighbor_ppos, int neighbor_count,
                                                scalar p0, scalar h);

// helper functions for getting grid indices
__device__ void kgrid_getGridLocation(Vector3s pos, int &i, int &j, int &k);
__device__ int kgrid_getGridIndex(int i, int j, int k);
__device__ void kgrid_getGridLocationFromIndex(int id, int &i, int &j, int &k);

// helper function for getting grid size
__host__ void hgrid_getGridSize(FluidBoundingBox* fbox, scalar h,
                               int &gridSizeX, int &gridSizeY, int &gridSizeZ);

// helper function for getting k nearest neighbors
__host__ void hgrid_findKNearestNeighbors(int **g_neighbors, int **g_gridIndex,
                                          int **g_grid,
                                          grid_gpu_block_t **g_particles,
                                          int num_particles,
                                          scalar h,
                                          int grid_size);

__device__ Vector3s kgrid_getFluidVolumePosition(FluidVolume& volume, int k);

__global__ void kgrid_initializePositions(grid_gpu_block_t *g_particles, FluidVolume* g_volumes,
                                     int num_particles, int num_volumes);
__global__ void kgrid_updateVBO(float* vbo, grid_gpu_block_t *g_particles, int num_particles);


// apply forces
__global__ void kgrid_applyForces(grid_gpu_block_t *g_particles,
                                  int num_particles,
                                  Vector3s accumForce,
                                  scalar dt);

// clear grid
__global__ void kgrid_clearGrid(int *g_grid, int grid_size);

// get grid indices
__global__ void kgrid_getGridIndices(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     int *g_gridIndex);

// get grid cells
__global__ void kgrid_setGridCells(int *g_gridIndex,
                                   int num_particles,
                                   int *g_grid,
                                   int num_cells);

// find k nearest neighbors
__global__ void kgrid_findKNearestNeighbors(grid_gpu_block_t *g_particles,
                                            int num_particles,
                                            int *g_neighbors,
                                            int *g_gridIndex,
                                            int *g_grid,
                                            int num_cells,
                                            scalar h);

// calculate lambda
__global__ void kgrid_calculateLambda(grid_gpu_block_t *g_particles,
                                      int num_particles,
                                      int *g_neighbors,
                                      scalar h,
                                      scalar p0);
// TODO


// update velocity
__global__ void kgrid_updateVelocity(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     scalar dt);

// update position
__global__ void kgrid_updatePosition(grid_gpu_block_t *g_particles,
                                     int num_particles);

// Nearest Neighbor kernels
__global__ void kgrid_clearGrid(int *grid, int grid_size);


////////////////////////////////////////////////
/// Implementation
////////////////////////////////////////////////

/// Init fluid

void grid_initGPUFluid(int **g_neighbors, int **g_gridIndex,
                       int **g_grid,
                       grid_gpu_block_t **g_particles,
                       FluidVolume* h_volumes, int num_volumes,
                       FluidBoundingBox* h_boundingBox,
                       scalar h) {

    int num_particles = 0;
    for (int i=0; i<num_volumes; i++) {
        num_particles += h_volumes[i].m_numParticles;
    }

    // initialize volumes array (free afterward)
    FluidVolume* g_volumes;
    GPU_CHECKERROR(cudaMalloc((void **)&g_volumes,
                              sizeof(FluidVolume)*num_volumes));
    // printf("%f, %f; %f, %f; %f, %f\n", h_volumes[0].m_minX, h_volumes[0].m_maxX, h_volumes[0].m_minY, h_volumes[0].m_maxY, h_volumes[0].m_minZ, h_volumes[0].m_maxZ); 

    GPU_CHECKERROR(cudaMemcpy((void *)g_volumes, (void *)h_volumes,
                              sizeof(FluidVolume)*num_volumes,
                              cudaMemcpyHostToDevice));

    // allocate particles
    GPU_CHECKERROR(cudaMalloc((void **)g_particles,
                              sizeof(grid_gpu_block_t)*num_particles));

    // allocate neighbors array (num_particles * num_neighbors * int)
    GPU_CHECKERROR(cudaMalloc((void **)g_neighbors,
                              sizeof(int)*num_particles*kgrid_NUM_NEIGHBORS));

    // allocate grid index array (num_particles * int)
    GPU_CHECKERROR(cudaMalloc((void **)g_gridIndex,
                              sizeof(int)*num_particles));


    int gridSize = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    kgrid_initializePositions <<< gridSize, kgrid_BLOCKSIZE_1D
                              >>> (*g_particles, g_volumes, num_particles, num_volumes);
    GPU_CHECKERROR(cudaGetLastError());

    //setup bounding box constants
    scalar h_minX = h_boundingBox->minX();
    scalar h_maxX = h_boundingBox->maxX();
    scalar h_minY = h_boundingBox->minY();
    scalar h_maxY = h_boundingBox->maxY();
    scalar h_minZ = h_boundingBox->minZ();
    scalar h_maxZ = h_boundingBox->maxZ();

    GPU_CHECKERROR(cudaMemcpyToSymbol(c_minX, &h_minX,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_maxX, &h_maxX,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_minY, &h_minY,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_maxY, &h_maxY,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_minZ, &h_minZ,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_maxZ, &h_maxZ,
                                      sizeof(scalar)));

    // figure out dimensions of grid
    int gridSizeX, gridSizeY, gridSizeZ;
    hgrid_getGridSize(h_boundingBox, h, gridSizeX, gridSizeY, gridSizeZ);

    GPU_CHECKERROR(cudaMemcpyToSymbol(c_gridSizeX, &gridSizeX,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_gridSizeY, &gridSizeY,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_gridSizeZ, &gridSizeZ,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_h, &h,
                                      sizeof(scalar)));

    // allocate grid
    GPU_CHECKERROR(cudaMalloc((void **)g_grid,
                              sizeof(int)*gridSizeX*gridSizeY*gridSizeZ));

    GPU_CHECKERROR(cudaThreadSynchronize());

    cudaFree(g_volumes); // we don't use this anymore
}

__global__ void kgrid_initializePositions(grid_gpu_block_t *g_particles, FluidVolume* g_volumes,
                                     int num_particles, int num_volumes) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= num_particles)
        return;

    int volume_index = -1;
    int offset = 0;
    int volume_size = 0;
    do {
        volume_index++;
        offset += volume_size;
        volume_size = g_volumes[volume_index].m_numParticles;
    } while (offset + volume_size < gid);

    FluidVolume& volume = g_volumes[volume_index];

    g_particles[gid].pos = kgrid_getFluidVolumePosition(volume, gid - offset);
    g_particles[gid].vec1 = Vector3s(0, 0, 0); // velocity
}

/// Update VBO

void grid_updateVBO(float *vboptr, grid_gpu_block_t *g_particles, int num_particles) {

    if (vboptr == NULL) {
        printf("oh no!!\n");
    }
    int gridSize = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    kgrid_updateVBO <<< gridSize, kgrid_BLOCKSIZE_1D
                    >>> (vboptr, g_particles, num_particles);

    cudaError_t err = cudaGetLastError();
    if(err != cudaSuccess){
        grid_deviceHappy = false;
        fprintf(stderr, "%s in %s at line %d\n", cudaGetErrorString(err), __FILE__, __LINE__);
        std::cout << "vboptr: " << vboptr << std::endl;
        return;
    }
    else {
        grid_deviceHappy = true;
    }

    GPU_CHECKERROR(cudaDeviceSynchronize());
}

__global__ void kgrid_updateVBO(float* vbo, grid_gpu_block_t *g_particles, int num_particles) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < num_particles) {
    //if(gid < 10){
        vbo[gid*4+0] = g_particles[gid].pos.x;
        vbo[gid*4+1] = g_particles[gid].pos.y;
        vbo[gid*4+2] = g_particles[gid].pos.z;
        //vbo[gid*4+0] = 5.0f;
        //vbo[gid*4+1] = 5.0f;
        //vbo[gid*4+2] = 5.0f;
        vbo[gid*4+3] = 1.0f;
    }
}

/// Step function

void grid_stepFluid(int **g_neighbors, int **g_gridIndex,
                    int **g_grid,
                    grid_gpu_block_t **g_particles,
                    int num_particles,
                    FluidBoundingBox* h_boundingBox,
                    scalar h,
                    scalar p0,
                    Vector3s accumForce,
                    scalar dt) {


    int gridSizeX, gridSizeY, gridSizeZ;
    hgrid_getGridSize(h_boundingBox, h, gridSizeX, gridSizeY, gridSizeZ);
    int grid_size = gridSizeX*gridSizeY*gridSizeZ;

    int blocksPerParticles = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    int blocksPerGridCells = ceil(grid_size / (kgrid_BLOCKSIZE_1D*1.0));
    int blocksPerPartReduced = ceil(num_particles / (kgrid_BLOCKSIZE_REDUCED*1.0));

    // step 1: apply forces, predict position
    kgrid_applyForces <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                      >>> (*g_particles, num_particles, accumForce, dt);
    GPU_CHECKERROR(cudaGetLastError());


    // step 2: find k nearest neighbors
    hgrid_findKNearestNeighbors(g_neighbors, g_gridIndex,
                                g_grid,
                                g_particles,
                                num_particles,
                                h,
                                grid_size);

    // step 3: calculate lambda
    size_t lambda_shared_bytes = sizeof(scalar) * kgrid_NUM_NEIGHBORS * kgrid_BLOCKSIZE_1D;
    kgrid_calculateLambda <<< blocksPerParticles, kgrid_BLOCKSIZE_1D, lambda_shared_bytes
                          >>> (*g_particles, num_particles,
                               *g_neighbors, h, p0);
    GPU_CHECKERROR(cudaGetLastError());

    // step 4: calculate dpos
    

    // TODO
    GPU_CHECKERROR(cudaDeviceSynchronize());

    // step 7: update velocity
    kgrid_updateVelocity <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                      >>> (*g_particles, num_particles, dt);
    GPU_CHECKERROR(cudaGetLastError());


    // step 9: update position
    kgrid_updatePosition <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                      >>> (*g_particles, num_particles);
    GPU_CHECKERROR(cudaGetLastError());



    GPU_CHECKERROR(cudaDeviceSynchronize());
}

// find k nearest neighbors

__host__ void hgrid_findKNearestNeighbors(int **g_neighbors, int **g_gridIndex,
                                          int **g_grid,
                                          grid_gpu_block_t **g_particles,
                                          int num_particles,
                                          scalar h,
                                          int grid_size) {
    int blocksPerParticles = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    int blocksPerGridCells = ceil(grid_size / (kgrid_BLOCKSIZE_1D*1.0));
    int blocksPerPartReduced = ceil(num_particles / (kgrid_BLOCKSIZE_REDUCED*1.0));

    // step 2a: reset grid
    kgrid_clearGrid <<< blocksPerGridCells, kgrid_BLOCKSIZE_1D
                    >>> (*g_grid, grid_size);
    GPU_CHECKERROR(cudaGetLastError());

    // step 2b: get gridIDs
    kgrid_getGridIndices <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                         >>> (*g_particles, num_particles, *g_gridIndex);
    GPU_CHECKERROR(cudaGetLastError());

    // step 2c: sort particles by gridID
    thrust::device_ptr<int> t_gridIndex = thrust::device_pointer_cast(*g_gridIndex);
    thrust::device_ptr<grid_gpu_block_t> t_particles =
        thrust::device_pointer_cast(*g_particles);

    thrust::sort_by_key(t_gridIndex, t_gridIndex+num_particles, t_particles);
    GPU_CHECKERROR(cudaGetLastError());

    // step 2d: set grid cells
    kgrid_setGridCells <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                       >>> (*g_gridIndex, num_particles, *g_grid, grid_size);
    GPU_CHECKERROR(cudaGetLastError());

    GPU_CHECKERROR(cudaDeviceSynchronize());

    // step 2f: find k nearest neighbors

    assert(sizeof(int) == sizeof(float));
    size_t knn_shared_bytes = sizeof(int) * kgrid_MAX_CELL_SIZE * kgrid_BLOCKSIZE_REDUCED * 2;
    kgrid_findKNearestNeighbors <<< blocksPerPartReduced, kgrid_BLOCKSIZE_REDUCED, knn_shared_bytes
                                >>> (*g_particles, num_particles, *g_neighbors,
                                     *g_gridIndex, *g_grid, grid_size, h);
    GPU_CHECKERROR(cudaGetLastError());
}

/// Apply Forces

__global__ void kgrid_applyForces(grid_gpu_block_t *g_particles,
                                  int num_particles,
                                  Vector3s accumForce,
                                  scalar dt) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        Vector3s pos = g_particles[id].pos;
        Vector3s vel = g_particles[id].vec1;
        vel += dt * accumForce;
        Vector3s ppos = pos + dt * vel;
        g_particles[id].vec1 = vel; //velocity
        g_particles[id].vec2 = ppos; // predicted pos
    }
}

/// Reset Grid

__global__ void kgrid_clearGrid(int *g_grid, int grid_size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < grid_size) {
        g_grid[id] = -1; // indicates no particle
    }
}

/// Get Grid Indices

__global__ void kgrid_getGridIndices(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     int *g_gridIndex) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        Vector3s pos = g_particles[id].pos;
        int i, j, k;
        kgrid_getGridLocation(pos, i, j, k);
        int index = kgrid_getGridIndex(i, j, k);
        g_gridIndex[id] = index;
    }
}

/// Set grid cells

__global__ void kgrid_setGridCells(int *g_gridIndex,
                                   int num_particles,
                                   int *g_grid,
                                   int num_cells) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        int gridCell = g_gridIndex[id];
        if (id == 0) {
            g_grid[gridCell] = id;
        }
        else if (gridCell != g_gridIndex[id - 1]) {
            if (gridCell < 0 || num_cells <= gridCell) {
                printf("INVALID GRIDCELL: %d\n", gridCell);
                return;
            }
            g_grid[gridCell] = id;
        }
    }
}

/// find k nearest neighbors (by particle)
__global__ void kgrid_findKNearestNeighbors(grid_gpu_block_t *g_particles,
                                            int num_particles,
                                            int *g_neighbors,
                                            int *g_gridIndex,
                                            int *g_grid,
                                            int num_cells,
                                            scalar h) {
    extern __shared__ int s_mem[]; // 32 bits for both float and int

    const int array_size = kgrid_MAX_CELL_SIZE * 2;
    const int thread_offset = array_size * threadIdx.x;
    const int local_offset = kgrid_MAX_CELL_SIZE;

    int* s_particles = &s_mem[thread_offset];
    float* s_distances = (float*)&s_particles[local_offset];

    // int s_particles[kgrid_MAX_CELL_SIZE];
    // float s_distances[kgrid_MAX_CELL_SIZE];

    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (particle_id >= num_particles)
        return;

    Vector3s pos = g_particles[particle_id].pos;
    int grid_index = g_gridIndex[particle_id];
    int i, j, k;
    kgrid_getGridLocationFromIndex(grid_index, i, j, k);

    int num_candidates = 0;
    for (int cur_i = i-1; cur_i <= i+1; cur_i++) {
        for (int cur_j = j-1; cur_j <= j+1; cur_j++) {
            for (int cur_k = k-1; cur_k <= k+1; cur_k++) {

                if (cur_i < 0 || cur_i >= c_gridSizeX)
                    continue;
                if (cur_j < 0 || cur_j >= c_gridSizeY)
                    continue;
                if (cur_k < 0 || cur_k >= c_gridSizeZ)
                    continue;
                if (num_candidates >= kgrid_MAX_CELL_SIZE)
                    goto nearest_postloop;

                int cur_grid_index = kgrid_getGridIndex(cur_i, cur_j, cur_k);

                int first_particle_id = g_grid[cur_grid_index];
                if (first_particle_id == -1)
                    continue;

                int cur_particle_id = first_particle_id;
                Vector3s cur_pos;
                scalar dist;

                while(cur_particle_id < num_particles &&
                      cur_grid_index != g_gridIndex[cur_particle_id]) {
                    // printf("while : %d - %d\n", particle_id, cur_particle_id);

                    if (cur_particle_id < 0 || cur_particle_id >= num_particles) {
                        printf("oh god oh god: %d - %d", particle_id, cur_particle_id);
                    }

                    cur_pos = g_particles[cur_particle_id].pos;


                    dist = glm::length(cur_pos - pos);

                    if (dist < h) {
                        s_particles[num_candidates] = cur_particle_id;
                        s_distances[num_candidates] = dist;

                        num_candidates++;
                    }
                    cur_particle_id++;
                }

            }
        }
    }

 nearest_postloop:
    // printf("sorting!\n");

    // now that the arrays are loaded, let's sort them
    thrust::device_ptr<int> t_particles = thrust::device_pointer_cast(s_particles);
    thrust::device_ptr<float> t_distances = thrust::device_pointer_cast(s_distances);

    thrust::stable_sort_by_key(thrust::device, t_distances, t_distances+num_candidates,
                        t_particles);

    int *g_my_neighbors = g_neighbors + (kgrid_NUM_NEIGHBORS * particle_id);

    // take first k particles and put them in neighbors list
    // put -1 if candidates don't exist
    for (int n_i = 0; n_i < kgrid_NUM_NEIGHBORS; n_i++) {
        if (n_i < num_candidates) {
            g_my_neighbors[n_i] = s_particles[n_i];
        }
        else {
            g_my_neighbors[n_i] = -1;
        }
    }
}

/// calculate lambda - by particle
/// input (vec2 = ppos) --> (sca1 = lambda)
__global__ void kgrid_calculateLambda(grid_gpu_block_t *g_particles,
                                      int num_particles,
                                      int *g_neighbors,
                                      scalar h,
                                      scalar p0) {
    extern __shared__ Vector3s s_neighbor_ppos[];

    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (particle_id >= num_particles)
        return;

    // copy over global memory to shared block
    // also keep track of actual neighbor count
    int neighbor_count;
    int *g_my_neighbors = g_neighbors + (kgrid_NUM_NEIGHBORS * particle_id);
    Vector3s *s_my_neighbor_ppos = s_neighbor_ppos + (kgrid_NUM_NEIGHBORS * threadIdx.x);

    //copy over data from global memory to shared memory
    for (neighbor_count=0; neighbor_count<kgrid_NUM_NEIGHBORS; neighbor_count++) {
        int neighbor_id = g_my_neighbors[neighbor_count];
        if (neighbor_id == -1)
            break;

        // copy over vec2 attribute (ppos)
        s_my_neighbor_ppos[neighbor_count] = g_particles[neighbor_id].vec2;
    }

    // get our own ppos
    Vector3s ppos = g_particles[particle_id].vec2;

    scalar press = 0;
    // iterate over neighbor array

    for (int i=0; i<neighbor_count; i++) {
        Vector3s &other_ppos = s_my_neighbor_ppos[i];
        press += kgrid_Poly6Kernel(ppos, other_ppos, h);
    }

    scalar top = (press / p0) - 1.0;

    // accumulate Ci gradients
    scalar gradSum = 0;
    scalar gradL;
    for (int i=0; i<neighbor_count; i++) {
        Vector3s &other_ppos = s_my_neighbor_ppos[i];
        gradL = glm::length(kgrid_calcGradConstraint(ppos, other_ppos, p0, h));
        gradSum = gradL*gradL;
    }
    //add self
    gradL = glm::length(kgrid_calcGradConstraintAtI(ppos,
                                                    s_my_neighbor_ppos,
                                                    neighbor_count,
                                                    p0, h));
    gradSum += gradL*gradL;
    gradSum += kgrid_RELAXATION;

    scalar lambda = -1.0f * top / gradSum;
    g_particles[particle_id].sca1 = lambda;
}

// Calculate dpos



/// TODO


/// update velocity

__global__ void kgrid_updateVelocity(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     scalar dt) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        Vector3s pos = g_particles[id].pos; // pos
        Vector3s ppos = g_particles[id].vec2; // ppos
        Vector3s vel = (ppos - pos) / dt;
        g_particles[id].vec1 = vel; //velocity
    }
}


/// update position

__global__ void kgrid_updatePosition(grid_gpu_block_t *g_particles,
                                     int num_particles) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        Vector3s ppos = g_particles[id].vec2; // ppos
        g_particles[id].pos = ppos; //pos = ppos
    }

}



////////////////////////////////////////
/// Helper functions
////////////////////////////////////////

__host__ void hgrid_getGridSize(FluidBoundingBox* fbox, scalar h,
                               int &gridSizeX, int &gridSizeY, int &gridSizeZ) {
    gridSizeX = ceil(fbox->width() / h);
    gridSizeY = ceil(fbox->height() / h);
    gridSizeZ = ceil(fbox->depth() / h);
}

__device__ void kgrid_getGridLocation(Vector3s pos, int &i, int &j, int &k) {

    scalar x = pos.x;
    scalar y = pos.y;
    scalar z = pos.z;

    i = (x - c_minX) / c_h;
    j = (y - c_minY) / c_h;
    k = (z - c_minZ) / c_h;
}

__device__ int kgrid_getGridIndex(int i, int j, int k) {
    return (c_gridSizeX * c_gridSizeY * k) + (c_gridSizeX * j) + i;
}

__device__ void kgrid_getGridLocationFromIndex(int id, int &i, int &j, int &k) {
    i = id % c_gridSizeX;
    j = (id / c_gridSizeX) % c_gridSizeY;
    k = (id / c_gridSizeX / c_gridSizeY) % c_gridSizeZ;
}

__device__ Vector3s kgrid_getFluidVolumePosition(FluidVolume& volume, int k) {

    if (volume.m_mode == kFLUID_VOLUME_MODE_BOX) {
        //random mode not supported
        int xlen = (volume.m_maxX - volume.m_minX) / volume.m_dens_cbrt;
        int ylen = (volume.m_maxY - volume.m_minY) / volume.m_dens_cbrt;
        int zlen = (volume.m_maxZ - volume.m_minZ) / volume.m_dens_cbrt;

        int xindex = (k / zlen / ylen) % xlen;
        int yindex = (k / zlen) % ylen;
        int zindex = k % zlen;

        // printf("%f - %d, %d, %d\n", volume.m_dens_cbrt, xindex, yindex, zindex);

        scalar x = xindex * volume.m_dens_cbrt;
        scalar y = yindex * volume.m_dens_cbrt;
        scalar z = zindex * volume.m_dens_cbrt;
        return Vector3s(x, y, z);
    }
    // sphere mode not supported
    return Vector3s(0, 0, 0);
}

///kernel functions

__device__ scalar kgrid_Poly6Kernel(Vector3s& pi, Vector3s& pj, scalar H){
    scalar r = glm::distance(pi, pj);
    if(r > H || r < 0)
        return 0;

    r = ((H * H) - (r * r));
    r = r * r * r; // (h^2 - r^2)^3
    return r * (315.0 / (64.0 * PI * H * H * H * H * H * H * H * H * H));

}

__device__ Vector3s kgrid_SpikyKernelGrad(Vector3s& pi, Vector3s& pj, scalar H){
    Vector3s dp = pi - pj;
    scalar r = glm::length(dp);
    if(r > H || r < 0)
        return Vector3s(0.0, 0.0, 0.0);
    scalar scale = 45.0 / (PI * H * H * H * H * H * H) * (H - r) * (H - r);
    return scale / (r + 0.001f) * dp;
}

// gradient functions

__device__ Vector3s kgrid_calcGradConstraint(Vector3s& pi, Vector3s& pj, scalar p0, scalar h){
    return -1.0f * kgrid_SpikyKernelGrad(pi, pj, h) / p0;
}

__device__ Vector3s kgrid_calcGradConstraintAtI(Vector3s &pi,
                                                Vector3s* neighbor_ppos, int neighbor_count,
                                                scalar p0, scalar h) {
    Vector3s sumGrad(0.0, 0.0, 0.0);

    for (int i=0; i<neighbor_count; i++) {
        Vector3s other_ppos = neighbor_ppos[i];
        sumGrad += kgrid_SpikyKernelGrad(pi, other_ppos, h);
    }

    return sumGrad / p0;
}


#endif
