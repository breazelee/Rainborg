
#include <hip/hip_runtime.h>
#ifdef GPU_ENABLED
#include "GridGPUFluidKernel.h"
#include "../FluidVolume.h"
#include "GPUHelper.h"

const int kgrid_BLOCKSIZE_1D = 512;
const int kgrid_BLOCKSIZE_3D = 8;

bool grid_deviceHappy = false;

__device__ Vector3s kgrid_getFluidVolumePosition(FluidVolume& volume, int k);

__global__ void kgrid_initializePositions(Vector3s *g_pos, FluidVolume* g_volumes,
                                     int num_particles, int num_volumes);
__global__ void kgrid_updateVBO(float* dptrvert, Vector3s *g_pos, int num_particles);

void grid_initGPUFluid(Vector3s *g_pos, Vector3s *g_vel,
                       int *g_neighbors, int *g_gridIndex,
                       FluidVolume* h_volumes, int num_volumes) {

    int num_particles = 0;
    for (int i=0; i<num_volumes; i++) {
        num_particles += h_volumes[i].m_numParticles;
    }

    // initialize volumes array (free afterward)
    FluidVolume* g_volumes;
    GPU_CHECKERROR(cudaMalloc((void **)&g_volumes,
                              sizeof(FluidVolume)*num_volumes));
    GPU_CHECKERROR(cudaMemcpy((void *)g_volumes, (void *)h_volumes,
                              sizeof(FluidVolume)*num_volumes,
                              cudaMemcpyHostToDevice));

    // allocate position and velocity array
    GPU_CHECKERROR(cudaMalloc((void **)&g_pos,
                              sizeof(Vector3s)*num_particles));
    GPU_CHECKERROR(cudaMalloc((void **)&g_vel,
                              sizeof(Vector3s)*num_particles));
    // set velocities to 0
    GPU_CHECKERROR(cudaMemset((void *)g_vel, 0,
                              sizeof(Vector3s)*num_particles));

    int gridSize = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    kgrid_initializePositions <<< gridSize, kgrid_BLOCKSIZE_1D
                              >>> (g_pos, g_volumes, num_particles, num_volumes);
    GPU_CHECKERROR(cudaGetLastError());
    GPU_CHECKERROR(cudaThreadSynchronize());

    cudaFree(g_volumes); // we don't use this anymore
}

void grid_updateVBO(float *vboptr, Vector3s *g_pos, int num_particles) {

    if (vboptr == NULL) {
        printf("oh no!!\n");
    }
    int gridSize = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    kgrid_updateVBO <<< gridSize, kgrid_BLOCKSIZE_1D
                    >>> (vboptr, g_pos, num_particles);

    cudaError_t err = cudaGetLastError();
    if(err != cudaSuccess){
        grid_deviceHappy = false;
        fprintf(stderr, "%s in %s at line %d\n", cudaGetErrorString(err), __FILE__, __LINE__);
        std::cout << "vboptr: " << vboptr << std::endl;
        return;
    }
    else {
        grid_deviceHappy = true;
    }

    GPU_CHECKERROR(cudaThreadSynchronize());
}

__global__ void kgrid_initializePositions(Vector3s *g_pos, FluidVolume* g_volumes,
                                     int num_particles, int num_volumes) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= num_particles)
        return;

    int volume_index = -1;
    int offset = 0;
    int volume_size = 0;
    do {
        volume_index++;
        offset += volume_size;
        volume_size = g_volumes[volume_index].m_numParticles;
    } while (offset + volume_size < gid);

    FluidVolume& volume = g_volumes[volume_index];
    g_pos[gid] = kgrid_getFluidVolumePosition(volume, gid - offset);
}

__global__ void kgrid_updateVBO(float* vbo, Vector3s *g_pos, int num_particles) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < num_particles) {
        vbo[gid*4+0] = g_pos[gid][0];
        vbo[gid*4+1] = g_pos[gid][1];
        vbo[gid*4+2] = g_pos[gid][2];
        vbo[gid*4+3] = 1.0f;
    }
}

__device__ Vector3s kgrid_getFluidVolumePosition(FluidVolume& volume, int k) {

    if (volume.m_mode == kFLUID_VOLUME_MODE_BOX) {
        //random mode not supported
        int xlen = (volume.m_maxX - volume.m_minX) / volume.m_dens_cbrt;
        int ylen = (volume.m_maxY - volume.m_minY) / volume.m_dens_cbrt;
        int zlen = (volume.m_maxZ - volume.m_minZ) / volume.m_dens_cbrt;

        int xindex = (k / zlen / ylen) % xlen;
        int yindex = (k / zlen) % ylen;
        int zindex = k % zlen;

        scalar x = xindex * volume.m_dens_cbrt;
        scalar y = yindex * volume.m_dens_cbrt;
        scalar z = zindex * volume.m_dens_cbrt;
        return Vector3s(x, y, z);
    }
    // sphere mode not supported
    return Vector3s(0, 0, 0);
}


#endif
