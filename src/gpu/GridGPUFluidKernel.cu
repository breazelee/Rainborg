
#include <hip/hip_runtime.h>
#ifdef GPU_ENABLED
#include "GridGPUFluidKernel.h"
#include "GPUHelper.h"
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/pair.h>
#include <thrust/execution_policy.h>

const int kgrid_BLOCKSIZE_1D = 512;
const int kgrid_BLOCKSIZE_REDUCED = 256;
const int kgrid_NUM_NEIGHBORS = 5;
const int kgrid_MAX_CELL_SIZE = 20;

bool grid_deviceHappy = true;

// fluid characteristics
__constant__ scalar c_h;

// grid bounds
__constant__ scalar c_minX;
__constant__ scalar c_maxX;
__constant__ scalar c_minY;
__constant__ scalar c_maxY;
__constant__ scalar c_minZ;
__constant__ scalar c_maxZ;
__constant__ scalar c_eps;

// grid size
__constant__ int c_gridSizeX;
__constant__ int c_gridSizeY;
__constant__ int c_gridSizeZ;

///////////////////////////////////////////////
/// Function Headers
///////////////////////////////////////////////

// helper functions for getting grid indices
__device__ void kgrid_getGridLocation(Vector3s pos, int &i, int &j, int &k);
__device__ int kgrid_getGridIndex(int i, int j, int k);
__device__ void kgrid_getGridLocationFromIndex(int id, int &i, int &j, int &k);

// helper function for getting grid size
__host__ void hgrid_getGridSize(FluidBoundingBox* fbox, scalar h,
                               int &gridSizeX, int &gridSizeY, int &gridSizeZ);

// helper function for getting k nearest neighbors
__host__ void hgrid_findKNearestNeighbors(int **g_neighbors, int **g_gridIndex,
                                          int **g_grid,
                                          int **g_gridUniqueIndex, int **g_partUniqueIndex,
                                          grid_gpu_block_t **g_particles,
                                          int num_particles,
                                          scalar h,
                                          int grid_size,
                                          int *grid_unique_size);

__device__ Vector3s kgrid_getFluidVolumePosition(FluidVolume& volume, int k);

__global__ void kgrid_initializePositions(grid_gpu_block_t *g_particles, FluidVolume* g_volumes,
                                     int num_particles, int num_volumes);
__global__ void kgrid_updateVBO(float* vbo, grid_gpu_block_t *g_particles, int num_particles);


// apply forces
__global__ void kgrid_applyForces(grid_gpu_block_t *g_particles,
                                  int num_particles,
                                  Vector3s accumForce,
                                  scalar dt);

// clear grid
__global__ void kgrid_clearGrid(int *g_grid, int grid_size);

// get grid indices
__global__ void kgrid_getGridIndices(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     int *g_gridIndex,
                                     int *g_partUniqueIndex);

// get grid cells
__global__ void kgrid_setGridCells(int *g_gridIndex,
                                   int num_particles,
                                   int *g_grid,
                                   int num_cells);

// find k nearest neighbors
__global__ void kgrid_findKNearestNeighbors(grid_gpu_block_t *g_particles,
                                            int num_particles,
                                            int *g_neighbors,
                                            int *g_gridIndex,
                                            int *g_grid,
                                            int num_cells,
                                            scalar h);

// TODO


// update velocity
__global__ void kgrid_updateVelocity(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     scalar dt);

// update position
__global__ void kgrid_updatePosition(grid_gpu_block_t *g_particles,
                                     int num_particles);

// Nearest Neighbor kernels
__global__ void kgrid_clearGrid(int *grid, int grid_size);


////////////////////////////////////////////////
/// Implementation
////////////////////////////////////////////////

/// Init fluid

void grid_initGPUFluid(int **g_neighbors, int **g_gridIndex,
                       int **g_grid,
                       int **g_gridUniqueIndex, int **g_partUniqueIndex,
                       grid_gpu_block_t **g_particles,
                       FluidVolume* h_volumes, int num_volumes,
                       FluidBoundingBox* h_boundingBox,
                       scalar h) {

    int num_particles = 0;
    for (int i=0; i<num_volumes; i++) {
        num_particles += h_volumes[i].m_numParticles;
    }

    // initialize volumes array (free afterward)
    FluidVolume* g_volumes;
    GPU_CHECKERROR(cudaMalloc((void **)&g_volumes,
                              sizeof(FluidVolume)*num_volumes));
    // printf("%f, %f; %f, %f; %f, %f\n", h_volumes[0].m_minX, h_volumes[0].m_maxX, h_volumes[0].m_minY, h_volumes[0].m_maxY, h_volumes[0].m_minZ, h_volumes[0].m_maxZ); 

    GPU_CHECKERROR(cudaMemcpy((void *)g_volumes, (void *)h_volumes,
                              sizeof(FluidVolume)*num_volumes,
                              cudaMemcpyHostToDevice));

    // allocate particles
    GPU_CHECKERROR(cudaMalloc((void **)g_particles,
                              sizeof(grid_gpu_block_t)*num_particles));

    // allocate neighbors array (num_particles * num_neighbors * int)
    GPU_CHECKERROR(cudaMalloc((void **)g_neighbors,
                              sizeof(int)*num_particles*kgrid_NUM_NEIGHBORS));

    // allocate grid index array (num_particles * int)
    GPU_CHECKERROR(cudaMalloc((void **)g_gridIndex,
                              sizeof(int)*num_particles));

    // allocate grid unique index array (initially num_particles * int)
    GPU_CHECKERROR(cudaMalloc((void **)g_gridUniqueIndex,
                              sizeof(int)*num_particles));

    // allocate part unique index array (initially num_particles * int)
    GPU_CHECKERROR(cudaMalloc((void **)g_partUniqueIndex,
                              sizeof(int)*num_particles));


    int gridSize = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    kgrid_initializePositions <<< gridSize, kgrid_BLOCKSIZE_1D
                              >>> (*g_particles, g_volumes, num_particles, num_volumes);
    GPU_CHECKERROR(cudaGetLastError());

    //setup bounding box constants
    scalar h_minX = h_boundingBox->minX();
    scalar h_maxX = h_boundingBox->maxX();
    scalar h_minY = h_boundingBox->minY();
    scalar h_maxY = h_boundingBox->maxY();
    scalar h_minZ = h_boundingBox->minZ();
    scalar h_maxZ = h_boundingBox->maxZ();

    GPU_CHECKERROR(cudaMemcpyToSymbol(c_minX, &h_minX,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_maxX, &h_maxX,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_minY, &h_minY,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_maxY, &h_maxY,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_minZ, &h_minZ,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_maxZ, &h_maxZ,
                                      sizeof(scalar)));

    // figure out dimensions of grid
    int gridSizeX, gridSizeY, gridSizeZ;
    hgrid_getGridSize(h_boundingBox, h, gridSizeX, gridSizeY, gridSizeZ);

    GPU_CHECKERROR(cudaMemcpyToSymbol(c_gridSizeX, &gridSizeX,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_gridSizeY, &gridSizeY,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_gridSizeZ, &gridSizeZ,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_h, &h,
                                      sizeof(scalar)));

    // allocate grid
    GPU_CHECKERROR(cudaMalloc((void **)g_grid,
                              sizeof(int)*gridSizeX*gridSizeY*gridSizeZ));

    GPU_CHECKERROR(cudaThreadSynchronize());

    cudaFree(g_volumes); // we don't use this anymore
}

__global__ void kgrid_initializePositions(grid_gpu_block_t *g_particles, FluidVolume* g_volumes,
                                     int num_particles, int num_volumes) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= num_particles)
        return;

    int volume_index = -1;
    int offset = 0;
    int volume_size = 0;
    do {
        volume_index++;
        offset += volume_size;
        volume_size = g_volumes[volume_index].m_numParticles;
    } while (offset + volume_size < gid);

    FluidVolume& volume = g_volumes[volume_index];

    g_particles[gid].pos = kgrid_getFluidVolumePosition(volume, gid - offset);
    g_particles[gid].vec1 = Vector3s(0, 0, 0); // velocity
}

/// Update VBO

void grid_updateVBO(float *vboptr, grid_gpu_block_t *g_particles, int num_particles) {

    if (vboptr == NULL) {
        printf("oh no!!\n");
    }
    int gridSize = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    kgrid_updateVBO <<< gridSize, kgrid_BLOCKSIZE_1D
                    >>> (vboptr, g_particles, num_particles);

    cudaError_t err = cudaGetLastError();
    if(err != cudaSuccess){
        grid_deviceHappy = false;
        fprintf(stderr, "%s in %s at line %d\n", cudaGetErrorString(err), __FILE__, __LINE__);
        std::cout << "vboptr: " << vboptr << std::endl;
        return;
    }
    else {
        grid_deviceHappy = true;
    }

    GPU_CHECKERROR(cudaDeviceSynchronize());
}

__global__ void kgrid_updateVBO(float* vbo, grid_gpu_block_t *g_particles, int num_particles) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < num_particles) {
    //if(gid < 10){
        vbo[gid*4+0] = g_particles[gid].pos.x;
        vbo[gid*4+1] = g_particles[gid].pos.y;
        vbo[gid*4+2] = g_particles[gid].pos.z;
        //vbo[gid*4+0] = 5.0f;
        //vbo[gid*4+1] = 5.0f;
        //vbo[gid*4+2] = 5.0f;
        vbo[gid*4+3] = 1.0f;
    }
}

/// Step function

void grid_stepFluid(int **g_neighbors, int **g_gridIndex,
                    int **g_grid,
                    int **g_gridUniqueIndex, int **g_partUniqueIndex,
                    grid_gpu_block_t **g_particles,
                    int num_particles,
                    FluidBoundingBox* h_boundingBox,
                    scalar h,
                    Vector3s accumForce,
                    scalar dt) {


    int gridSizeX, gridSizeY, gridSizeZ;
    hgrid_getGridSize(h_boundingBox, h, gridSizeX, gridSizeY, gridSizeZ);
    int grid_size = gridSizeX*gridSizeY*gridSizeZ;

    int blocksPerParticles = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    int blocksPerGridCells = ceil(grid_size / (kgrid_BLOCKSIZE_1D*1.0));

    // step 1: apply forces, predict position
    kgrid_applyForces <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                      >>> (*g_particles, num_particles, accumForce, dt);
    GPU_CHECKERROR(cudaGetLastError());


    // step 2: find k nearest neighbors
    int grid_unique_size;
    hgrid_findKNearestNeighbors(g_neighbors, g_gridIndex,
                                g_grid,
                                g_gridUniqueIndex, g_partUniqueIndex,
                                g_particles,
                                num_particles,
                                h,
                                grid_size,
                                &grid_unique_size);

    GPU_CHECKERROR(cudaDeviceSynchronize());



    // TODO

    // step 7: update velocity
    kgrid_updateVelocity <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                      >>> (*g_particles, num_particles, dt);
    GPU_CHECKERROR(cudaGetLastError());


    // step 9: update position
    kgrid_updatePosition <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                      >>> (*g_particles, num_particles);
    GPU_CHECKERROR(cudaGetLastError());



    GPU_CHECKERROR(cudaDeviceSynchronize());
}

// find k nearest neighbors

__host__ void hgrid_findKNearestNeighbors(int **g_neighbors, int **g_gridIndex,
                                          int **g_grid,
                                          int **g_gridUniqueIndex, int **g_partUniqueIndex,
                                          grid_gpu_block_t **g_particles,
                                          int num_particles,
                                          scalar h,
                                          int grid_size,
                                          int *grid_unique_size) {
    int blocksPerParticles = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    int blocksPerGridCells = ceil(grid_size / (kgrid_BLOCKSIZE_1D*1.0));
    int blocksPerPartReduced = ceil(num_particles / (kgrid_BLOCKSIZE_REDUCED*1.0));

    // step 2a: reset grid
    kgrid_clearGrid <<< blocksPerGridCells, kgrid_BLOCKSIZE_1D
                    >>> (*g_grid, grid_size);
    GPU_CHECKERROR(cudaGetLastError());

    // step 2b: get gridIDs
    kgrid_getGridIndices <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                         >>> (*g_particles, num_particles, *g_gridIndex, *g_partUniqueIndex);
    GPU_CHECKERROR(cudaGetLastError());

    // step 2c: sort particles by gridID
    thrust::device_ptr<int> t_gridIndex = thrust::device_pointer_cast(*g_gridIndex);
    thrust::device_ptr<grid_gpu_block_t> t_particles =
        thrust::device_pointer_cast(*g_particles);

    thrust::sort_by_key(t_gridIndex, t_gridIndex+num_particles, t_particles);
    GPU_CHECKERROR(cudaGetLastError());

    // step 2d: set grid cells
    kgrid_setGridCells <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                       >>> (*g_gridIndex, num_particles, *g_grid, grid_size);
    GPU_CHECKERROR(cudaGetLastError());

    // step 2e: find unique grid cells / particles, find number unique cells
    thrust::device_ptr<int> t_gridUniqueIndex =
        thrust::device_pointer_cast(*g_gridUniqueIndex);
    thrust::device_ptr<int> t_partUniqueIndex =
        thrust::device_pointer_cast(*g_partUniqueIndex);

    // copy over to grid unique index
    thrust::pair<thrust::device_ptr<int>, thrust::device_ptr<int> > t_unique_end =
        thrust::unique_by_key_copy(thrust::device,
                                   t_gridIndex,
                                   t_gridIndex + num_particles,
                                   t_partUniqueIndex,
                                   t_gridUniqueIndex,
                                   t_partUniqueIndex);

    GPU_CHECKERROR(cudaDeviceSynchronize());
    *grid_unique_size = t_unique_end.first - t_gridUniqueIndex;

    // std::cout << "unique size is: " << grid_unique_size << std::endl;
    // std::cout << "grid size is : " << grid_size << std::endl;
    // step 2f: find k nearest neighbors

    assert(sizeof(int) == sizeof(float));
    size_t knn_shared_bytes = sizeof(int) * kgrid_MAX_CELL_SIZE * kgrid_BLOCKSIZE_REDUCED * 2;
    kgrid_findKNearestNeighbors <<< blocksPerPartReduced, kgrid_BLOCKSIZE_REDUCED, knn_shared_bytes
                                >>> (*g_particles, num_particles, *g_neighbors,
                                     *g_gridIndex, *g_grid, grid_size, h);
    GPU_CHECKERROR(cudaGetLastError());
}

/// Apply Forces

__global__ void kgrid_applyForces(grid_gpu_block_t *g_particles,
                                  int num_particles,
                                  Vector3s accumForce,
                                  scalar dt) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        Vector3s pos = g_particles[id].pos;
        Vector3s vel = g_particles[id].vec1;
        vel += dt * accumForce;
        Vector3s ppos = pos + dt * vel;
        g_particles[id].vec1 = vel; //velocity
        g_particles[id].vec2 = ppos; // predicted pos
    }
}

/// Reset Grid

__global__ void kgrid_clearGrid(int *g_grid, int grid_size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < grid_size) {
        g_grid[id] = -1; // indicates no particle
    }
}

/// Get Grid Indices (also setup unique particle index)

__global__ void kgrid_getGridIndices(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     int *g_gridIndex,
                                     int *g_partUniqueIndex) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        Vector3s pos = g_particles[id].pos;
        int i, j, k;
        kgrid_getGridLocation(pos, i, j, k);
        int index = kgrid_getGridIndex(i, j, k);
        g_gridIndex[id] = index;
        g_partUniqueIndex[id] = id;
    }
}

/// Set grid cells

__global__ void kgrid_setGridCells(int *g_gridIndex,
                                   int num_particles,
                                   int *g_grid,
                                   int num_cells) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        int gridCell = g_gridIndex[id];
        if (id == 0) {
            g_grid[gridCell] = id;
        }
        else if (gridCell != g_gridIndex[id - 1]) {
            if (gridCell < 0 || num_cells <= gridCell) {
                printf("INVALID GRIDCELL: %d\n", gridCell);
                return;
            }
            g_grid[gridCell] = id;
        }
    }
}

/// find k nearest neighbors (by particle)
__global__ void kgrid_findKNearestNeighbors(grid_gpu_block_t *g_particles,
                                            int num_particles,
                                            int *g_neighbors,
                                            int *g_gridIndex,
                                            int *g_grid,
                                            int num_cells,
                                            scalar h) {
    extern __shared__ int s_mem[]; // 32 bits for both float and int

    const int array_size = kgrid_MAX_CELL_SIZE * 2;
    const int thread_offset = array_size * threadIdx.x;
    const int local_offset = kgrid_MAX_CELL_SIZE;

    int* s_particles = &s_mem[thread_offset];
    float* s_distances = (float*)&s_particles[local_offset];

    // int s_particles[kgrid_MAX_CELL_SIZE];
    // float s_distances[kgrid_MAX_CELL_SIZE];

    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (particle_id >= num_particles)
        return;

    Vector3s pos = g_particles[particle_id].pos;
    int grid_index = g_gridIndex[particle_id];
    int i, j, k;
    kgrid_getGridLocationFromIndex(grid_index, i, j, k);

    int num_candidates = 0;
    for (int cur_i = i-1; cur_i <= i+1; cur_i++) {
        for (int cur_j = j-1; cur_j <= j+1; cur_j++) {
            for (int cur_k = k-1; cur_k <= k+1; cur_k++) {

                if (cur_i < 0 || cur_i >= c_gridSizeX)
                    continue;
                if (cur_j < 0 || cur_j >= c_gridSizeY)
                    continue;
                if (cur_k < 0 || cur_k >= c_gridSizeZ)
                    continue;
                if (num_candidates >= kgrid_MAX_CELL_SIZE)
                    goto nearest_postloop;

                int cur_grid_index = kgrid_getGridIndex(cur_i, cur_j, cur_k);

                int first_particle_id = g_grid[cur_grid_index];
                if (first_particle_id == -1)
                    continue;

                int cur_particle_id = first_particle_id;
                Vector3s cur_pos;
                scalar dist;

                while(cur_particle_id < num_particles &&
                      cur_grid_index != g_gridIndex[cur_particle_id]) {
                    // printf("while : %d - %d\n", particle_id, cur_particle_id);

                    if (cur_particle_id < 0 || cur_particle_id >= num_particles) {
                        printf("oh god oh god: %d - %d", particle_id, cur_particle_id);
                    }

                    cur_pos = g_particles[cur_particle_id].pos;


                    dist = glm::length(cur_pos - pos);

                    if (dist < h) {
                        s_particles[num_candidates] = cur_particle_id;
                        s_distances[num_candidates] = dist;

                        num_candidates++;
                    }
                    cur_particle_id++;
                }

            }
        }
    }

 nearest_postloop:
    // printf("sorting!\n");

    // now that the arrays are loaded, let's sort them
    thrust::device_ptr<int> t_particles = thrust::device_pointer_cast(s_particles);
    thrust::device_ptr<float> t_distances = thrust::device_pointer_cast(s_distances);

    thrust::stable_sort_by_key(thrust::seq, t_distances, t_distances+num_candidates,
                        t_particles);

    int *g_my_neighbors = g_neighbors + (kgrid_NUM_NEIGHBORS * particle_id);

    // take first k particles and put them in neighbors list
    // put -1 if candidates don't exist
    for (int n_i = 0; n_i < kgrid_NUM_NEIGHBORS; n_i++) {
        if (n_i < num_candidates) {
            g_my_neighbors[n_i] = s_particles[n_i];
        }
        else {
            g_my_neighbors[n_i] = -1;
        }
    }
}


/// TODO


/// update velocity

__global__ void kgrid_updateVelocity(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     scalar dt) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        Vector3s pos = g_particles[id].pos; // pos
        Vector3s ppos = g_particles[id].vec2; // ppos
        Vector3s vel = (ppos - pos) / dt;
        g_particles[id].vec1 = vel; //velocity
    }
}


/// update position

__global__ void kgrid_updatePosition(grid_gpu_block_t *g_particles,
                                     int num_particles) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        Vector3s ppos = g_particles[id].vec2; // ppos
        g_particles[id].pos = ppos; //pos = ppos
    }

}



////////////////////////////////////////
/// Helper functions
////////////////////////////////////////

__host__ void hgrid_getGridSize(FluidBoundingBox* fbox, scalar h,
                               int &gridSizeX, int &gridSizeY, int &gridSizeZ) {
    gridSizeX = ceil(fbox->width() / h);
    gridSizeY = ceil(fbox->height() / h);
    gridSizeZ = ceil(fbox->depth() / h);
}

__device__ void kgrid_getGridLocation(Vector3s pos, int &i, int &j, int &k) {

    scalar x = pos.x;
    scalar y = pos.y;
    scalar z = pos.z;

    i = (x - c_minX) / c_h;
    j = (y - c_minY) / c_h;
    k = (z - c_minZ) / c_h;
}

__device__ int kgrid_getGridIndex(int i, int j, int k) {
    return (c_gridSizeX * c_gridSizeY * k) + (c_gridSizeX * j) + i;
}

__device__ void kgrid_getGridLocationFromIndex(int id, int &i, int &j, int &k) {
    i = id % c_gridSizeX;
    j = (id / c_gridSizeX) % c_gridSizeY;
    k = (id / c_gridSizeX / c_gridSizeY) % c_gridSizeZ;
}

__device__ Vector3s kgrid_getFluidVolumePosition(FluidVolume& volume, int k) {

    if (volume.m_mode == kFLUID_VOLUME_MODE_BOX) {
        //random mode not supported
        int xlen = (volume.m_maxX - volume.m_minX) / volume.m_dens_cbrt;
        int ylen = (volume.m_maxY - volume.m_minY) / volume.m_dens_cbrt;
        int zlen = (volume.m_maxZ - volume.m_minZ) / volume.m_dens_cbrt;

        int xindex = (k / zlen / ylen) % xlen;
        int yindex = (k / zlen) % ylen;
        int zindex = k % zlen;

        // printf("%f - %d, %d, %d\n", volume.m_dens_cbrt, xindex, yindex, zindex);

        scalar x = xindex * volume.m_dens_cbrt;
        scalar y = yindex * volume.m_dens_cbrt;
        scalar z = zindex * volume.m_dens_cbrt;
        return Vector3s(x, y, z);
    }
    // sphere mode not supported
    return Vector3s(0, 0, 0);
}


#endif
