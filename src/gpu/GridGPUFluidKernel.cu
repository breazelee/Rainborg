
#include <hip/hip_runtime.h>
#ifdef GPU_ENABLED
#include "GridGPUFluidKernel.h"
#include "GPUHelper.h"
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/pair.h>
#include <thrust/execution_policy.h>

#define GRID_ART_PRESSURE 1
#define GRID_VORTICITY 1

const int kgrid_BLOCKSIZE_1D = 512;
const int kgrid_BLOCKSIZE_REDUCED = 256;
const int kgrid_NUM_NEIGHBORS = 5;
const int kgrid_MAX_CELL_SIZE = 20;

const scalar kgrid_EPS = 0.001;

const scalar kgrid_RELAXATION = 0.01;

// artificial pressure constants
const scalar kgrid_DELTA_Q_SCALE = 0.1;
const scalar kgrid_ART_PRESSURE_K = 0.1;
const scalar kgrid_ART_PRESSURE_N = 6;

// XSPH constants
const scalar kgrid_XSPH_C = 0.01;
const scalar kgrid_VORTICITY_EPS = 0.01;

bool grid_deviceHappy = true;

// fluid characteristics
__constant__ scalar c_h;

// grid bounds
__constant__ scalar c_minX;
__constant__ scalar c_maxX;
__constant__ scalar c_minY;
__constant__ scalar c_maxY;
__constant__ scalar c_minZ;
__constant__ scalar c_maxZ;
__constant__ scalar c_eps;

// grid size
__constant__ int c_gridSizeX;
__constant__ int c_gridSizeY;
__constant__ int c_gridSizeZ;

///////////////////////////////////////////////
/// Function Headers
///////////////////////////////////////////////

// kernel functions
__device__ scalar kgrid_Poly6Kernel(Vector3s &pi, Vector3s &pj, scalar h);
__device__ Vector3s kgrid_SpikyKernelGrad(Vector3s &pi, Vector3s &pj, scalar h);

// gradient functions
__device__ Vector3s kgrid_calcGradConstraint(Vector3s& pi, Vector3s& pj, scalar p0, scalar h);
__device__ Vector3s kgrid_calcGradConstraintAtI(Vector3s &pi,
                                                Vector3s* neighbor_ppos, int neighbor_count,
                                                scalar p0, scalar h);

// helper functions for getting grid indices
__device__ void kgrid_getGridLocation(Vector3s pos, int &i, int &j, int &k);
__device__ int kgrid_getGridIndex(int i, int j, int k);
__device__ void kgrid_getGridLocationFromIndex(int id, int &i, int &j, int &k);

// helper function for getting grid size
__host__ void hgrid_getGridSize(FluidBoundingBox* fbox, scalar h,
                               int &gridSizeX, int &gridSizeY, int &gridSizeZ);

// helper function for getting k nearest neighbors
__host__ void hgrid_findKNearestNeighbors(int **g_neighbors, int **g_gridIndex,
                                          int **g_grid,
                                          grid_gpu_block_t **g_particles,
                                          int num_particles,
                                          scalar h,
                                          int grid_size);

__device__ Vector3s kgrid_getFluidVolumePosition(FluidVolume& volume, int k);

__global__ void kgrid_initializePositions(grid_gpu_block_t *g_particles, FluidVolume* g_volumes,
                                     int num_particles, int num_volumes);
__global__ void kgrid_updateVBO(float* vbo, grid_gpu_block_t *g_particles, int num_particles);


// apply forces
__global__ void kgrid_applyForces(grid_gpu_block_t *g_particles,
                                  int num_particles,
                                  Vector3s accumForce,
                                  scalar mass,
                                  scalar dt);

// clear grid
__global__ void kgrid_clearGrid(int *g_grid, int grid_size);

// get grid indices
__global__ void kgrid_getGridIndices(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     int *g_gridIndex);

// get grid cells
__global__ void kgrid_setGridCells(int *g_gridIndex,
                                   int num_particles,
                                   int *g_grid,
                                   int num_cells);

// find k nearest neighbors
__global__ void kgrid_findKNearestNeighbors(grid_gpu_block_t *g_particles,
                                            int num_particles,
                                            int *g_neighbors,
                                            int *g_gridIndex,
                                            int *g_grid,
                                            int num_cells,
                                            scalar h);

// calculate lambda
__global__ void kgrid_calculateLambda(grid_gpu_block_t *g_particles,
                                      int num_particles,
                                      int *g_neighbors,
                                      scalar mass,
                                      scalar h,
                                      scalar p0);

// calculate dpos
__global__ void kgrid_calculateDPos(grid_gpu_block_t *g_particles,
                                    int num_particles,
                                    int *g_neighbors,
                                    scalar h, scalar p0);

// preserve fluid boundary
__global__ void kgrid_preserveFluidBoundary(grid_gpu_block_t *g_particles,
                                            int num_particles);

// update ppos
__global__ void kgrid_updatePPos(grid_gpu_block_t *g_particles,
                                 int num_particles);


// update velocity
__global__ void kgrid_updateVelocity(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     scalar dt);

// apply xsph and viscosity
__global__ void kgrid_applyXSPHAndOmega(grid_gpu_block_t *g_particles,
                                        int num_particles,
                                        int *g_neighbors,
                                        scalar h);

// apply vorticity
__global__ void kgrid_applyVorticity(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     int *g_neighbors,
                                     scalar h);

// update position
__global__ void kgrid_updatePosition(grid_gpu_block_t *g_particles,
                                     int num_particles);

// Nearest Neighbor kernels
__global__ void kgrid_clearGrid(int *grid, int grid_size);


////////////////////////////////////////////////
/// Implementation
////////////////////////////////////////////////

/// Init fluid

void grid_initGPUFluid(int **g_neighbors, int **g_gridIndex,
                       int **g_grid,
                       grid_gpu_block_t **g_particles,
                       FluidVolume* h_volumes, int num_volumes,
                       FluidBoundingBox* h_boundingBox,
                       scalar h) {

    int num_particles = 0;
    for (int i=0; i<num_volumes; i++) {
        num_particles += h_volumes[i].m_numParticles;
    }

    // initialize volumes array (free afterward)
    FluidVolume* g_volumes;
    GPU_CHECKERROR(cudaMalloc((void **)&g_volumes,
                              sizeof(FluidVolume)*num_volumes));
    // printf("%f, %f; %f, %f; %f, %f\n", h_volumes[0].m_minX, h_volumes[0].m_maxX, h_volumes[0].m_minY, h_volumes[0].m_maxY, h_volumes[0].m_minZ, h_volumes[0].m_maxZ); 

    GPU_CHECKERROR(cudaMemcpy((void *)g_volumes, (void *)h_volumes,
                              sizeof(FluidVolume)*num_volumes,
                              cudaMemcpyHostToDevice));

    // allocate particles
    GPU_CHECKERROR(cudaMalloc((void **)g_particles,
                              sizeof(grid_gpu_block_t)*num_particles));

    // allocate neighbors array (num_particles * num_neighbors * int)
    GPU_CHECKERROR(cudaMalloc((void **)g_neighbors,
                              sizeof(int)*num_particles*kgrid_NUM_NEIGHBORS));

    // allocate grid index array (num_particles * int)
    GPU_CHECKERROR(cudaMalloc((void **)g_gridIndex,
                              sizeof(int)*num_particles));


    int gridSize = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    kgrid_initializePositions <<< gridSize, kgrid_BLOCKSIZE_1D
                              >>> (*g_particles, g_volumes, num_particles, num_volumes);
    GPU_CHECKERROR(cudaGetLastError());

    //setup bounding box constants
    scalar h_minX = h_boundingBox->minX();
    scalar h_maxX = h_boundingBox->maxX();
    scalar h_minY = h_boundingBox->minY();
    scalar h_maxY = h_boundingBox->maxY();
    scalar h_minZ = h_boundingBox->minZ();
    scalar h_maxZ = h_boundingBox->maxZ();

    GPU_CHECKERROR(cudaMemcpyToSymbol(c_minX, &h_minX,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_maxX, &h_maxX,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_minY, &h_minY,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_maxY, &h_maxY,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_minZ, &h_minZ,
                                      sizeof(scalar)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_maxZ, &h_maxZ,
                                      sizeof(scalar)));

    // figure out dimensions of grid
    int gridSizeX, gridSizeY, gridSizeZ;
    hgrid_getGridSize(h_boundingBox, h, gridSizeX, gridSizeY, gridSizeZ);

    GPU_CHECKERROR(cudaMemcpyToSymbol(c_gridSizeX, &gridSizeX,
                                      sizeof(int)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_gridSizeY, &gridSizeY,
                                      sizeof(int)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_gridSizeZ, &gridSizeZ,
                                      sizeof(int)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(c_h, &h,
                                      sizeof(scalar)));

    // allocate grid
    GPU_CHECKERROR(cudaMalloc((void **)g_grid,
                              sizeof(int)*gridSizeX*gridSizeY*gridSizeZ));

    GPU_CHECKERROR(cudaThreadSynchronize());

    cudaFree(g_volumes); // we don't use this anymore
}

__global__ void kgrid_initializePositions(grid_gpu_block_t *g_particles, FluidVolume* g_volumes,
                                     int num_particles, int num_volumes) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= num_particles)
        return;

    int volume_index = -1;
    int offset = 0;
    int volume_size = 0;
    do {
        volume_index++;
        offset += volume_size;
        volume_size = g_volumes[volume_index].m_numParticles;
    } while (offset + volume_size < gid);

    FluidVolume& volume = g_volumes[volume_index];

    g_particles[gid].pos = kgrid_getFluidVolumePosition(volume, gid - offset);
    g_particles[gid].vec1 = Vector3s(0, 0, 0); // velocity
    g_particles[gid].vec3 = Vector3s(0,0,0); //ext-force

    // load colors
    g_particles[gid].r = volume.m_color.r * 255.0f;
    g_particles[gid].g = volume.m_color.g * 255.0f;
    g_particles[gid].b = volume.m_color.b * 255.0f;
    g_particles[gid].a = volume.m_color.a * 255.0f;

    g_particles[gid].num_neighbors = 0;
}

/// Update VBO

void grid_updateVBO(float *vboptr, grid_gpu_block_t *g_particles, int num_particles) {

    if (vboptr == NULL) {
        printf("oh no!!\n");
    }
    int gridSize = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    kgrid_updateVBO <<< gridSize, kgrid_BLOCKSIZE_1D
                    >>> (vboptr, g_particles, num_particles);

    cudaError_t err = cudaGetLastError();
    if(err != cudaSuccess){
        grid_deviceHappy = false;
        fprintf(stderr, "%s in %s at line %d\n", cudaGetErrorString(err), __FILE__, __LINE__);
        std::cout << "vboptr: " << vboptr << std::endl;
        return;
    }
    else {
        grid_deviceHappy = true;
    }

    GPU_CHECKERROR(cudaDeviceSynchronize());
}

__global__ void kgrid_updateVBO(float* vbo, grid_gpu_block_t *g_particles, int num_particles) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < num_particles) {
    //if(gid < 10){
        grid_gpu_block_t my_particle = g_particles[gid];
        vbo[gid*4+0] = my_particle.pos.x;
        vbo[gid*4+1] = my_particle.pos.y;
        vbo[gid*4+2] = my_particle.pos.z;
        char *colors = (char *)&vbo[gid*4+3];
        scalar depth = (my_particle.pos.y - c_minY) / (c_maxY - c_minY);
        scalar num_neighbors = my_particle.num_neighbors;
        num_neighbors = kgrid_MAX_CELL_SIZE - num_neighbors - 10; // 5 to 0
        if (num_neighbors < 0) num_neighbors = 0;
        num_neighbors /= kgrid_MAX_CELL_SIZE - 10; // 1 to 0
        colors[0] = (char)(9 + (num_neighbors * (65-9)));
        colors[1] = (char)(24 + (num_neighbors * (191-24)));
        colors[2] = (char)(84 + (num_neighbors * (229-84)));
        colors[3] = (char)178;
    }
}

/// Step function

void grid_stepFluid(int **g_neighbors, int **g_gridIndex,
                    int **g_grid,
                    grid_gpu_block_t **g_particles,
                    int num_particles,
                    FluidBoundingBox* h_boundingBox,
                    int iters,
                    scalar mass,
                    scalar h,
                    scalar p0,
                    Vector3s accumForce,
                    scalar dt) {


    int gridSizeX, gridSizeY, gridSizeZ;
    hgrid_getGridSize(h_boundingBox, h, gridSizeX, gridSizeY, gridSizeZ);
    int grid_size = gridSizeX*gridSizeY*gridSizeZ;

    int blocksPerParticles = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    int blocksPerGridCells = ceil(grid_size / (kgrid_BLOCKSIZE_1D*1.0));
    int blocksPerPartReduced = ceil(num_particles / (kgrid_BLOCKSIZE_REDUCED*1.0));

    // step 1: apply forces, predict position
    kgrid_applyForces <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                      >>> (*g_particles, num_particles, accumForce, mass, dt);
    GPU_CHECKERROR(cudaGetLastError());

    // step 2: find k nearest neighbors
    hgrid_findKNearestNeighbors(g_neighbors, g_gridIndex,
                                g_grid,
                                g_particles,
                                num_particles,
                                h,
                                grid_size);
    GPU_CHECKERROR(cudaGetLastError());

    for (int i = 0; i< iters; i++) {

        // step 3: calculate lambda
        size_t lambda_shared_bytes = sizeof(Vector3s) * kgrid_NUM_NEIGHBORS * kgrid_BLOCKSIZE_1D;
        kgrid_calculateLambda <<< blocksPerParticles, kgrid_BLOCKSIZE_1D, lambda_shared_bytes
                              >>> (*g_particles, num_particles,
                                   *g_neighbors, mass, h, p0);
        GPU_CHECKERROR(cudaGetLastError());

        // step 4: calculate dpos
        kgrid_calculateDPos <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                            >>> (*g_particles, num_particles,
                                 *g_neighbors, h, p0);
        GPU_CHECKERROR(cudaGetLastError());

        // step 5: preserve fluid boundary
        kgrid_preserveFluidBoundary <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                                    >>> (*g_particles, num_particles);

        GPU_CHECKERROR(cudaGetLastError());

        // step 6: update ppos
        kgrid_updatePPos <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                         >>> (*g_particles, num_particles);
        GPU_CHECKERROR(cudaGetLastError());

    }


    // step 7: update velocity
    kgrid_updateVelocity <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                      >>> (*g_particles, num_particles, dt);
    GPU_CHECKERROR(cudaGetLastError());


    // step 8a: apply XSPH and set omega
    kgrid_applyXSPHAndOmega <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                            >>> (*g_particles, num_particles,
                                 *g_neighbors, h);
    GPU_CHECKERROR(cudaGetLastError());


    // step 8b: apply vorticity
    kgrid_applyVorticity <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                         >>> (*g_particles, num_particles,
                              *g_neighbors, h);
    GPU_CHECKERROR(cudaGetLastError());


    // step 9: update position
    kgrid_updatePosition <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                      >>> (*g_particles, num_particles);
    GPU_CHECKERROR(cudaGetLastError());


    GPU_CHECKERROR(cudaDeviceSynchronize());
}

// find k nearest neighbors

__host__ void hgrid_findKNearestNeighbors(int **g_neighbors, int **g_gridIndex,
                                          int **g_grid,
                                          grid_gpu_block_t **g_particles,
                                          int num_particles,
                                          scalar h,
                                          int grid_size) {
    int blocksPerParticles = ceil(num_particles / (kgrid_BLOCKSIZE_1D*1.0));
    int blocksPerGridCells = ceil(grid_size / (kgrid_BLOCKSIZE_1D*1.0));
    int blocksPerPartReduced = ceil(num_particles / (kgrid_BLOCKSIZE_REDUCED*1.0));

    // step 2a: reset grid
    kgrid_clearGrid <<< blocksPerGridCells, kgrid_BLOCKSIZE_1D
                    >>> (*g_grid, grid_size);
    GPU_CHECKERROR(cudaGetLastError());

    // step 2b: get gridIDs
    kgrid_getGridIndices <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                         >>> (*g_particles, num_particles, *g_gridIndex);
    GPU_CHECKERROR(cudaGetLastError());

    // step 2c: sort particles by gridID
    thrust::device_ptr<int> t_gridIndex = thrust::device_pointer_cast(*g_gridIndex);
    thrust::device_ptr<grid_gpu_block_t> t_particles =
        thrust::device_pointer_cast(*g_particles);

    thrust::sort_by_key(t_gridIndex, t_gridIndex+num_particles, t_particles);
    GPU_CHECKERROR(cudaGetLastError());

    // step 2d: set grid cells
    kgrid_setGridCells <<< blocksPerParticles, kgrid_BLOCKSIZE_1D
                       >>> (*g_gridIndex, num_particles, *g_grid, grid_size);
    GPU_CHECKERROR(cudaGetLastError());


    // step 2f: find k nearest neighbors

    assert(sizeof(int) == sizeof(float));
    size_t knn_shared_bytes = sizeof(int) * kgrid_MAX_CELL_SIZE * kgrid_BLOCKSIZE_REDUCED * 2;
    kgrid_findKNearestNeighbors <<< blocksPerPartReduced, kgrid_BLOCKSIZE_REDUCED, knn_shared_bytes
                                >>> (*g_particles, num_particles, *g_neighbors,
                                     *g_gridIndex, *g_grid, grid_size, h);
    GPU_CHECKERROR(cudaGetLastError());
}

/// Apply Forces

__global__ void kgrid_applyForces(grid_gpu_block_t *g_particles,
                                  int num_particles,
                                  Vector3s accumForce,
                                  scalar mass,
                                  scalar dt) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        Vector3s pos = g_particles[id].pos;
        Vector3s vel = g_particles[id].vec1;
        Vector3s ext_force = g_particles[id].vec3;
        vel += dt * (accumForce + ext_force) / mass;
        Vector3s ppos = pos + dt * vel;
        g_particles[id].vec1 = vel; //velocity
        g_particles[id].vec2 = ppos; // predicted pos
    }
}

/// Reset Grid

__global__ void kgrid_clearGrid(int *g_grid, int grid_size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < grid_size) {
        g_grid[id] = -1; // indicates no particle
    }
}

/// Get Grid Indices

__global__ void kgrid_getGridIndices(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     int *g_gridIndex) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        Vector3s pos = g_particles[id].pos;
        int i, j, k;
        kgrid_getGridLocation(pos, i, j, k);
        int index = kgrid_getGridIndex(i, j, k);
        g_gridIndex[id] = index;
    }
}

/// Set grid cells

__global__ void kgrid_setGridCells(int *g_gridIndex,
                                   int num_particles,
                                   int *g_grid,
                                   int num_cells) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        int gridCell = g_gridIndex[id];
        if (id == 0) {
            g_grid[gridCell] = id;
        }
        else if (gridCell != g_gridIndex[id - 1]) {
            if (gridCell < 0 || num_cells <= gridCell) {
                return;
            }
            g_grid[gridCell] = id;
        }
    }
}

/// find k nearest neighbors (by particle)
/// shared memory size: max_cell_size * 2 * blockSize * sizeof(int)
__global__ void kgrid_findKNearestNeighbors(grid_gpu_block_t *g_particles,
                                            int num_particles,
                                            int *g_neighbors,
                                            int *g_gridIndex,
                                            int *g_grid,
                                            int num_cells,
                                            scalar h) {
    extern __shared__ int s_mem[]; // 32 bits for both float and int

    const int array_size = kgrid_MAX_CELL_SIZE * 2;
    const int thread_offset = array_size * threadIdx.x;
    const int local_offset = kgrid_MAX_CELL_SIZE;

    int* s_particles = s_mem + thread_offset;
    float* s_distances = (float*)(s_particles + local_offset);

    // int s_particles[kgrid_MAX_CELL_SIZE];
    // float s_distances[kgrid_MAX_CELL_SIZE];

    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (particle_id >= num_particles)
        return;

    // if (particle_id != 0)
        // return;

    Vector3s pos = g_particles[particle_id].vec2;
    int grid_index = g_gridIndex[particle_id];
    int i, j, k;
    kgrid_getGridLocationFromIndex(grid_index, i, j, k);

    int num_candidates = 0;
    for (int cur_i = i-1; cur_i <= i+1; cur_i++) {
        for (int cur_j = j-1; cur_j <= j+1; cur_j++) {
            for (int cur_k = k-1; cur_k <= k+1; cur_k++) {

                if (cur_i < 0 || cur_i >= c_gridSizeX)
                    continue;
                if (cur_j < 0 || cur_j >= c_gridSizeY)
                    continue;
                if (cur_k < 0 || cur_k >= c_gridSizeZ)
                    continue;
                if (num_candidates >= kgrid_MAX_CELL_SIZE)
                    goto nearest_postloop;

                int cur_grid_index = kgrid_getGridIndex(cur_i, cur_j, cur_k);

                int first_particle_id = g_grid[cur_grid_index];
                if (first_particle_id == -1)
                    continue;

                int cur_particle_id = first_particle_id;
                Vector3s cur_pos;
                scalar dist;

                scalar maxDist = 0;
                int max = 0;

                while(cur_particle_id < num_particles &&
                      cur_grid_index == g_gridIndex[cur_particle_id]) {

                    // printf("while : %d - %d\n", particle_id, cur_particle_id);

                    if (cur_particle_id < 0 || cur_particle_id >= num_particles) {
                        printf("oh god oh god: %d - %d", particle_id, cur_particle_id);
                    }

                    cur_pos = g_particles[cur_particle_id].vec2;

                    dist = glm::length(cur_pos - pos);

                    if (dist < h) {

                        if (num_candidates >= kgrid_MAX_CELL_SIZE) {

                            // don't increment num_candidates
                            // check for maxDist
                            if (dist < maxDist) {
                                // we deserve a spot! replace id with
                                // max.
                                maxDist = dist;


                                s_particles[max] = cur_particle_id;
                                s_distances[max] = dist;

                                // recalculate maxdist
                                for (int r=0; r<kgrid_MAX_CELL_SIZE; r++) {
                                    scalar my_dist = s_distances[r];
                                    if (my_dist > maxDist) {
                                        maxDist = my_dist;
                                        max = r;
                                    }
                                }
                            }
                        }
                        else {

                            s_particles[num_candidates] = cur_particle_id;
                            s_distances[num_candidates] = dist;

                            //update max dist
                            if (dist > maxDist) {
                                maxDist = dist;
                                max = num_candidates;
                            }

                            num_candidates++;

                        }

                    }
                    cur_particle_id++;
                }

            }
        }
    }

 nearest_postloop:
    // printf("sorting!\n");

    // now that the arrays are loaded, let's sort them
    thrust::device_ptr<int> t_particles = thrust::device_pointer_cast(s_particles);
    thrust::device_ptr<float> t_distances = thrust::device_pointer_cast(s_distances);

    thrust::sort_by_key(thrust::seq, t_distances, t_distances+num_candidates,
                        t_particles);

    int *g_my_neighbors = g_neighbors + (kgrid_NUM_NEIGHBORS * particle_id);

    // take first k particles and put them in neighbors list
    // put -1 if candidates don't exist
    Vector3s delta(0,0,0);

    for (int n_i = 0; n_i < kgrid_NUM_NEIGHBORS; n_i++) {
        if (n_i < num_candidates) {
            g_my_neighbors[n_i] = s_particles[n_i];

            Vector3s other_pos = g_particles[s_particles[n_i]].vec2;
            delta += (other_pos - pos);

        }
        else {
            g_my_neighbors[n_i] = -1;
        }
    }

    delta /= num_candidates;
    delta *= 100.0f;
    g_particles[particle_id].num_neighbors = num_candidates;
}

/// calculate lambda - by particle
/// input (vec2 = ppos) --> (sca1 = lambda)
/// s_mem size: num_neighbors * block_size * size(Vector3s)
__global__ void kgrid_calculateLambda(grid_gpu_block_t *g_particles,
                                      int num_particles,
                                      int *g_neighbors,
                                      scalar mass,
                                      scalar h,
                                      scalar p0) {
    extern __shared__ Vector3s s_neighbor_ppos[];

    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (particle_id >= num_particles)
        return;

    // note that this is a copy of a struct. Don't output to this!
    grid_gpu_block_t my_particle = g_particles[particle_id];

    // copy over global memory to shared block
    // also keep track of actual neighbor count
    int neighbor_count;
    int *g_my_neighbors = g_neighbors + (kgrid_NUM_NEIGHBORS * particle_id);
    Vector3s *s_my_neighbor_ppos = &s_neighbor_ppos[kgrid_NUM_NEIGHBORS * threadIdx.x];

    //copy over data from global memory to shared memory
    for (neighbor_count=0; neighbor_count<kgrid_NUM_NEIGHBORS; neighbor_count++) {
        int neighbor_id = g_my_neighbors[neighbor_count];
        if (neighbor_id == -1)
            break;

        // copy over vec2 attribute (ppos)
        s_my_neighbor_ppos[neighbor_count] = g_particles[neighbor_id].vec2;
    }

    // get our own ppos
    Vector3s ppos = my_particle.vec2;

    scalar press = 0;
    // iterate over neighbor array

    for (int i=0; i<neighbor_count; i++) {
        Vector3s &other_ppos = s_my_neighbor_ppos[i];
        press += kgrid_Poly6Kernel(ppos, other_ppos, h);
    }
    press *= mass;
    if (neighbor_count == 0) {
        press = p0;
    }

    scalar top = (press / p0) - 1.0;

    // accumulate Ci gradients
    scalar gradSum = 0;
    scalar gradL;
    for (int i=0; i<neighbor_count; i++) {
        Vector3s &other_ppos = s_my_neighbor_ppos[i];
        gradL = glm::length(kgrid_calcGradConstraint(ppos, other_ppos, p0, h));
        gradSum = gradL*gradL;
    }
    //add self
    gradL = glm::length(kgrid_calcGradConstraintAtI(ppos,
                                                    s_my_neighbor_ppos,
                                                    neighbor_count,
                                                    p0, h));
    gradSum += gradL*gradL;
    gradSum += kgrid_RELAXATION;

    scalar lambda = -1.0f * top / gradSum;
    g_particles[particle_id].sca1 = lambda;
}

// Calculate dpos
// (vec2 = ppos, sca1 = lambda) --> (vec3 = dpos)
__global__ void kgrid_calculateDPos(grid_gpu_block_t *g_particles,
                                    int num_particles,
                                    int *g_neighbors,
                                    scalar h, scalar p0) {

    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (particle_id >= num_particles)
        return;

    // note that this is a copy of a struct. Don't output to this!
    grid_gpu_block_t my_particle = g_particles[particle_id];

    int *g_my_neighbors = g_neighbors + (kgrid_NUM_NEIGHBORS * particle_id);

    // get our own ppos and lambda
    Vector3s ppos = my_particle.vec2;
    scalar lambda = my_particle.sca1;

    Vector3s dp(0, 0, 0);
#if GRID_ART_PRESSURE == 1
    // some distance from position
    Vector3s q = kgrid_DELTA_Q_SCALE*h * glm::vec3(1.0f) + ppos;
#endif

    scalar scorr = 0;
    for (int i=0; i<kgrid_NUM_NEIGHBORS; i++) {
        int neighbor_id = g_my_neighbors[i];
        if (neighbor_id == -1)
            break;

        grid_gpu_block_t other_particle = g_particles[neighbor_id];
        Vector3s other_ppos = other_particle.vec2;
        scalar other_lambda = other_particle.sca1;

#if GRID_ART_PRESSURE == 1
        scalar top = kgrid_Poly6Kernel(ppos, other_ppos, h);
        scalar dq_kernel = kgrid_Poly6Kernel(ppos, q, h);
        scorr = -1.0f * kgrid_ART_PRESSURE_K * pow(top/dq_kernel, kgrid_ART_PRESSURE_N);
#endif
        dp += (other_lambda + lambda + scorr) * kgrid_SpikyKernelGrad(ppos, other_ppos, h);
    }

    g_particles[particle_id].vec3 = dp / p0; // delta pos mapped to vec3
}

// preserve fluid boundary
__global__ void kgrid_preserveFluidBoundary(grid_gpu_block_t *g_particles,
                                            int num_particles) {
    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (particle_id >= num_particles)
        return;

    // needless hack to randomize epsilon
    scalar shift = (particle_id%124) * 1.0 / 10000.0;

    grid_gpu_block_t my_particle = g_particles[particle_id];
    Vector3s ppos = my_particle.vec2;
    Vector3s dpos = my_particle.vec3;
    Vector3s pos = ppos + dpos;
    scalar posX = pos.x;
    scalar posY = pos.y;
    scalar posZ = pos.z;

    if (posX < c_minX)
        posX = c_minX + kgrid_EPS + shift;
    else if (posX > c_maxX)
        posX = c_maxX - kgrid_EPS - shift;
    if (posY < c_minY)
        posY = c_minY + kgrid_EPS + shift;
    else if (posY > c_maxY)
        posY = c_maxY - kgrid_EPS - shift;
    if (posZ < c_minZ)
        posZ = c_minZ + kgrid_EPS + shift;
    else if (posZ > c_maxZ)
        posZ = c_maxZ - kgrid_EPS - shift;

    g_particles[particle_id].vec3 = Vector3s(posX, posY, posZ) - ppos;
}

// update ppos
__global__ void kgrid_updatePPos(grid_gpu_block_t *g_particles,
                                 int num_particles) {
    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (particle_id >= num_particles)
        return;

    grid_gpu_block_t my_particle = g_particles[particle_id];
    Vector3s ppos = my_particle.vec2;
    Vector3s dpos = my_particle.vec3;

    ppos += dpos;

    g_particles[particle_id].vec2 = ppos;
}


/// update velocity

__global__ void kgrid_updateVelocity(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     scalar dt) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        Vector3s pos = g_particles[id].pos; // pos
        Vector3s ppos = g_particles[id].vec2; // ppos
        Vector3s vel = (ppos - pos) / dt;
        g_particles[id].vec1 = vel; //velocity
    }
}

// apply xsph and viscosity
// (vec2 = ppos, vec1 = vel) --> (pos = vel, vec3 = omega)
__global__ void kgrid_applyXSPHAndOmega(grid_gpu_block_t *g_particles,
                                        int num_particles,
                                        int *g_neighbors,
                                        scalar h) {
    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (particle_id >= num_particles)
        return;

    grid_gpu_block_t my_particle = g_particles[particle_id];
    Vector3s vel = my_particle.vec1;
    Vector3s ppos = my_particle.vec2;

    Vector3s dv(0,0,0);
#if GRID_VORTICITY == 1
    Vector3s omega(0,0,0);
#endif
    int *g_my_neighbors = g_neighbors + (kgrid_NUM_NEIGHBORS * particle_id);

    for (int i=0; i<kgrid_NUM_NEIGHBORS; i++) {
        int neighbor_id = g_my_neighbors[i];
        if (neighbor_id == -1)
            break;

        grid_gpu_block_t other_particle = g_particles[neighbor_id];
        Vector3s other_vel = other_particle.vec1;
        Vector3s other_ppos = other_particle.vec2;
        Vector3s vij = other_vel - vel;

        dv += vij * kgrid_Poly6Kernel(ppos, other_ppos, h);
#if GRID_VORTICITY == 1
        omega += glm::cross(vij, kgrid_SpikyKernelGrad(ppos, other_ppos, h));
#endif
    }

    dv *= kgrid_XSPH_C;
    vel += dv;
    g_particles[particle_id].pos = vel;

#if GRID_VORTICITY == 1
    g_particles[particle_id].vec3 = omega;
#endif
}

// apply vorticity
// (pos = vel, vec2 = ppos, vec3 = omega) -> (vec1 = vel, vec3 = ext force?)
__global__ void kgrid_applyVorticity(grid_gpu_block_t *g_particles,
                                     int num_particles,
                                     int *g_neighbors,
                                     scalar h) {
    int particle_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (particle_id >= num_particles)
        return;

    grid_gpu_block_t my_particle = g_particles[particle_id];
    Vector3s ppos = my_particle.vec2;
    Vector3s vel = my_particle.pos;
    Vector3s omega = my_particle.vec3;

    Vector3s vort(0,0,0);
    Vector3s grad(0,0,0);

    int *g_my_neighbors = g_neighbors + (kgrid_NUM_NEIGHBORS * particle_id);

    for (int i=0; i<kgrid_NUM_NEIGHBORS; i++) {
        int neighbor_id = g_my_neighbors[i];
        if (neighbor_id == -1)
            break;

        grid_gpu_block_t other_particle = g_particles[neighbor_id];
        Vector3s other_ppos = other_particle.vec2;
        Vector3s other_vel = other_particle.pos;
        Vector3s other_omega = other_particle.vec3;

        scalar dom = glm::length(omega - other_omega);
        Vector3s dp = other_ppos - ppos;
        vort.x += dom / (dp.x+0.001);
        vort.y += dom / (dp.y+0.001);
        vort.z += dom / (dp.z+0.001);
    }

    vort /= (glm::length(vort) + kgrid_EPS);

    g_particles[particle_id].vec1 = vel;

#if GRID_VORTICITY == 1
    Vector3s ext_force = 1.0f * kgrid_VORTICITY_EPS * glm::cross(vort, omega);
    g_particles[particle_id].vec3 = ext_force;
#else
    g_particles[particle_id].vec3 = Vector3s(0,0,0);
#endif
}

/// update position

__global__ void kgrid_updatePosition(grid_gpu_block_t *g_particles,
                                     int num_particles) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_particles) {
        Vector3s ppos = g_particles[id].vec2; // ppos
        g_particles[id].pos = ppos; //pos = ppos
    }

}



////////////////////////////////////////
/// Helper functions
////////////////////////////////////////

__host__ void hgrid_getGridSize(FluidBoundingBox* fbox, scalar h,
                               int &gridSizeX, int &gridSizeY, int &gridSizeZ) {
    gridSizeX = ceil(fbox->width() / h);
    gridSizeY = ceil(fbox->height() / h);
    gridSizeZ = ceil(fbox->depth() / h);
}

__device__ void kgrid_getGridLocation(Vector3s pos, int &i, int &j, int &k) {

    scalar x = pos.x;
    scalar y = pos.y;
    scalar z = pos.z;

    i = (x - c_minX) / c_h;
    j = (y - c_minY) / c_h;
    k = (z - c_minZ) / c_h;
}

__device__ int kgrid_getGridIndex(int i, int j, int k) {
    return (c_gridSizeX * c_gridSizeY * k) + (c_gridSizeX * j) + i;
}

__device__ void kgrid_getGridLocationFromIndex(int id, int &i, int &j, int &k) {
    i = id % c_gridSizeX;
    j = (id / c_gridSizeX) % c_gridSizeY;
    k = (id / c_gridSizeX / c_gridSizeY) % c_gridSizeZ;
}

__device__ Vector3s kgrid_getFluidVolumePosition(FluidVolume& volume, int k) {

    if (volume.m_mode == kFLUID_VOLUME_MODE_BOX) {

        //random mode not supported
        int xlen = (volume.m_maxX - volume.m_minX) / volume.m_dens_cbrt;
        int ylen = (volume.m_maxY - volume.m_minY) / volume.m_dens_cbrt;
        int zlen = (volume.m_maxZ - volume.m_minZ) / volume.m_dens_cbrt;

        int xindex = (k / zlen / ylen) % xlen;
        int yindex = (k / zlen) % ylen;
        int zindex = k % zlen;

        // add small epsilon to semi-randomize particles
        scalar xeps = volume.m_dens_cbrt*0.01 * ((yindex+zindex)%2);
        scalar yeps = volume.m_dens_cbrt*0.01 * ((xindex+zindex)%2);
        scalar zeps = volume.m_dens_cbrt*0.01 * ((xindex+yindex)%2);

        // printf("%f - %d, %d, %d\n", volume.m_dens_cbrt, xindex, yindex, zindex);

        scalar x = xindex * volume.m_dens_cbrt + xeps;
        scalar y = yindex * volume.m_dens_cbrt + yeps;
        scalar z = zindex * volume.m_dens_cbrt + zeps;
        return Vector3s(x, y, z);
    }
    // sphere mode not supported
    return Vector3s(0, 0, 0);
}

///kernel functions

__device__ scalar kgrid_Poly6Kernel(Vector3s& pi, Vector3s& pj, scalar H){
    scalar r = glm::distance(pi, pj);
    if(r > H || r < 0)
        return 0;

    r = ((H * H) - (r * r));
    r = r * r * r; // (h^2 - r^2)^3
    return r * (315.0 / (64.0 * PI * H * H * H * H * H * H * H * H * H));

}

__device__ Vector3s kgrid_SpikyKernelGrad(Vector3s& pi, Vector3s& pj, scalar H){
    Vector3s dp = pi - pj;
    scalar r = glm::length(dp);
    if(r > H || r < 0)
        return Vector3s(0.0, 0.0, 0.0);
    scalar scale = 45.0 / (PI * H * H * H * H * H * H) * (H - r) * (H - r);
    return scale / (r + 0.001f) * dp;
}

// gradient functions

__device__ Vector3s kgrid_calcGradConstraint(Vector3s& pi, Vector3s& pj, scalar p0, scalar h){
    return -1.0f * kgrid_SpikyKernelGrad(pi, pj, h) / p0;
}

__device__ Vector3s kgrid_calcGradConstraintAtI(Vector3s &pi,
                                                Vector3s* neighbor_ppos, int neighbor_count,
                                                scalar p0, scalar h) {
    Vector3s sumGrad(0.0, 0.0, 0.0);

    for (int i=0; i<neighbor_count; i++) {
        Vector3s other_ppos = neighbor_ppos[i];
        sumGrad += kgrid_SpikyKernelGrad(pi, other_ppos, h);
    }

    return sumGrad / p0;
}


#endif
