
#include <hip/hip_runtime.h>
#ifdef GPU_ENABLED
#include "GPUFluid.h"

#define BLOCKSIZE 256

bool deviceHappy = false; 

#define GPU_CHECKERROR(err) (gpuCheckError(err, __FILE__, __LINE__))
static void gpuCheckError(cudaError_t err, const char *file, int line){
    if(err != cudaSuccess){
        fprintf(stderr, "%s in %s at line %d\n", cudaGetErrorString(err), file, line);
    }   
}


__constant__ int GRIDX; 
__constant__ int GRIDY;
__constant__ int GRIDZ;
__constant__ scalar QSCALE; // for artificial pressure term 

Vector3s *d_pos;
Vector3s *d_vel; 
Vector3s *d_ppos;
Vector3s *d_dpos;
Vector3s *d_omega = NULL; 
scalar *d_pcalc; 
scalar *d_lambda; 

int *d_grid;
int *d_gridCount;
int *d_gridInd; 

int grid_X;
int grid_Y;
int grid_Z;

__device__ __host__ scalar wPoly6Kernel(Vector3s pi, Vector3s pj){
    scalar r = glm::distance(pi, pj); 
    if(r > H || r < 0)
        return 0; 

    r = ((H * H) - (r * r)); 
    r = r * r * r; // (h^2 - r^2)^3
    return r * (315.0 / (64.0 * PI * H * H * H * H * H * H * H * H * H));

}

__device__ __host__ Vector3s wSpikyKernelGrad(Vector3s pi, Vector3s pj){
    Vector3s dp = pi - pj; 
    scalar r = glm::length(dp);  
    if(r > H || r < 0)
        return Vector3s(0.0, 0.0, 0.0); 
    scalar scale = -45.0 / (PI * H * H * H * H * H * H) * (H - r) * (H - r); 
    return scale * dp; 
}

__device__ void getGridIdx(Vector3s pos, int* i, int *j, int *k){
    *i = (pos[0] - XMIN)/H;
    *j = (pos[1] - YMIN)/H;
    *k = (pos[2] - ZMIN)/H;
}

__device__ int get1DGridIdx(int i, int j, int k){
    return GRIDX * GRIDY * k + GRIDX * j + i;
}



__device__ Vector3s calcGradConstraint(Vector3s pi, Vector3s pj){
    return wSpikyKernelGrad(pi, pj)/(scalar(- P0)); 
}

__device__ Vector3s calcGradConstraintAtI(int p, Vector3s* d_ppos, int *d_grid, int *d_gridCount, int *d_gridInd){
    Vector3s sumGrad(0.0, 0.0, 0.0);
    Vector3s pi = d_ppos[p]; 
    int gi; 
    for(int i = max(0, d_gridInd[p*3]-1); i <= min(GRIDX-1, d_gridInd[p*3]+1); ++i){
        for(int j = max(0, d_gridInd[p*3+1]-1); j <= min(GRIDY-1, d_gridInd[p*3+1]+1); ++j){
            for(int k = max(0, d_gridInd[p*3+2]-1); k <= min(GRIDZ-1, d_gridInd[p*3+2]+1); ++k){
                gi = get1DGridIdx(i, j, k);
                for(int n = 0; n < d_gridCount[gi]; ++n){ // for all particles in the grid
                    sumGrad += wSpikyKernelGrad(pi, d_ppos[d_grid[gi * MAX_NEIGHBORS + n]]);
                }
            }
        }
    }     
    return sumGrad / (scalar)P0; 

}

__global__ void sendToVBO(float *vbo, Vector3s* d_pos){
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(id < NUM_PARTICLES){
        vbo[id*4+0] = d_pos[id][0];
        vbo[id*4+1] = d_pos[id][1];
        vbo[id*4+2] = d_pos[id][2];
        vbo[id*4+3] = 1.0f;
        //vbo[id*4+0] = 1.0f; 
        //vbo[id*4+1] = 1.0f;
        //vbo[id*4+2] = 1.0f;
        //vbo[id*4+3] = 1.0f;

    }

}

__global__ void updateFromForce(Vector3s* d_pos, Vector3s* d_vel, Vector3s* d_ppos, scalar dt, Vector3s force){
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(id < NUM_PARTICLES){
        d_vel[id] += force * dt / ((scalar)FP_MASS);
        //d_vel[id] += force * dt;
        d_ppos[id] = d_pos[id] + d_vel[id]*dt; 
    }
}

__global__ void applydPToPPos(Vector3s* d_ppos, Vector3s* d_dpos){
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(id < NUM_PARTICLES){
        d_ppos[id] += d_dpos[id];
        //d_ppos[id] += Vector3s(1.0, 0, 0);
    }
}

__global__ void preserveFluidBoundaryWithUpdate(Vector3s* d_pos, Vector3s* d_ppos, Vector3s* d_dpos){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(i >= NUM_PARTICLES){
        return;
    }
    scalar pposX = d_ppos[i][0] + d_dpos[i][0];
    scalar pposY = d_ppos[i][1] + d_dpos[i][1];
    scalar pposZ = d_ppos[i][2] + d_dpos[i][2]; 

    if(pposX < XMIN + EPS){
        d_dpos[i][0] = XMIN + EPS - d_ppos[i][0];
        d_ppos[i][0] += d_dpos[i][0];
    }
    else if(pposX > XMAX - EPS){
        d_dpos[i][0] = XMAX - EPS - d_ppos[i][0]; 
        d_ppos[i][0] += d_dpos[i][0];
    }
    if(pposY < YMIN + EPS){
        d_dpos[i][1] = YMIN + EPS - d_ppos[i][1];
        d_ppos[i][1] += d_dpos[i][1];
    }
    else if(pposY > YMAX - EPS){
        d_dpos[i][1] = YMAX - EPS - d_ppos[i][1];
        d_ppos[i][1] += d_dpos[i][1];
    }
    if(pposZ < ZMIN + EPS){
        d_dpos[i][2] = ZMIN + EPS - d_ppos[i][2];
        d_ppos[i][2] += d_dpos[i][2];
    }
    else if(pposZ > ZMAX - EPS){
        d_dpos[i][2] = ZMAX - EPS - d_ppos[i][2];
        d_ppos[i][2] += d_dpos[i][2];
    }
}

__global__ void preserveFluidBoundary(Vector3s *d_pos, Vector3s *d_ppos, Vector3s *d_dpos){
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(i >= NUM_PARTICLES){
        return;
    }
    scalar pposX = d_ppos[i][0] + d_dpos[i][0];
    scalar pposY = d_ppos[i][1] + d_dpos[i][1];
    scalar pposZ = d_ppos[i][2] + d_dpos[i][2]; 

    if(pposX < XMIN + EPS){
        d_dpos[i][0] = XMIN + EPS - d_ppos[i][0];
    }
    else if(pposX > XMAX - EPS){
        d_dpos[i][0] = XMAX - EPS - d_ppos[i][0]; 
    }
    if(pposY < YMIN + EPS){
        d_dpos[i][1] = YMIN + EPS - d_ppos[i][1];
    }
    else if(pposY > YMAX - EPS){
        d_dpos[i][1] = YMAX - EPS - d_ppos[i][1];
    }
    if(pposZ < ZMIN + EPS){
        d_dpos[i][2] = ZMIN + EPS - d_ppos[i][2];
    }
    else if(pposZ > ZMAX - EPS){
        d_dpos[i][2] = ZMAX - EPS - d_ppos[i][2];
    }
}

__global__ void buildGrid(Vector3s *d_ppos, int *d_grid, int *d_gridCount, int *d_gridInd){
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(id >= NUM_PARTICLES){
        return;
    }
    
    int gx; 
    int gy; 
    int gz; 
    getGridIdx(d_ppos[id], &gx, &gy, &gz);
    int gid = get1DGridIdx(gx, gy, gz);
    d_gridInd[id * 3] = gx;
    d_gridInd[id * 3+1] = gy;
    d_gridInd[id * 3+2] = gz;

    int actgid = gid * MAX_NEIGHBORS + d_gridCount[gid];
      
    bool placed = false;
    while(!placed){
        int result = atomicCAS(&(d_grid[actgid]), -1, id);
        if(result == -1){
            placed = true;
        }
        else{
            actgid ++;
        }
    }
    //d_grid[gid * MAX_NEIGHBORS + d_gridCount[gid]] = id;
    atomicAdd(&d_gridCount[gid], 1);
}

__global__ void calcPressures(Vector3s *d_ppos, int *d_grid, int *d_gridCount, int *d_gridInd, scalar *d_pcalc){
    int p = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(p >= NUM_PARTICLES){
        return;
    }
    scalar press = 0;
    int ncount = 0;
    int gi;
    for(int i = max(0, d_gridInd[p*3]-1); i <= min(GRIDX-1, d_gridInd[p*3]+1); ++i){
        for(int j = max(0, d_gridInd[p*3+1]-1); j <= min(GRIDY-1, d_gridInd[p*3+1]+1); ++j){
            for(int k = max(0, d_gridInd[p*3+2]-1); k <= min(GRIDZ-1, d_gridInd[p*3+2]+1); ++k){
                gi = get1DGridIdx(i, j, k);
                for(int n = 0; n < d_gridCount[gi]; ++n){ // for all particles in the grid
                    scalar pressN = wPoly6Kernel(d_ppos[p], d_ppos[d_grid[gi * MAX_NEIGHBORS + n]]); 
                    press += pressN;
                    if(pressN > 0)
                        ++ ncount; 

                }
            }
        }
    }     
    if(ncount <= MIN_NEIGHBORS && d_pcalc[p] == 0) // don't count self
        d_pcalc[p] = P0; 
    else 
        d_pcalc[p] = FP_MASS * press; // Wow I totally forgot that

}

__global__ void calcLambdas(Vector3s *d_ppos, int *d_grid, int *d_gridCount, int *d_gridInd, scalar *d_pcalc, scalar *d_lambda){
    int p = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(p >= NUM_PARTICLES)
        return;

    scalar top = -(d_pcalc[p]/P0 - 1.0);
    scalar gradSum = 0;
    scalar gradL = 0;
    Vector3s pi = d_ppos[p];

    int gi;
    for(int i = max(0, d_gridInd[p*3]-1); i <= min(GRIDX-1, d_gridInd[p*3]+1); ++i){
        for(int j = max(0, d_gridInd[p*3+1]-1); j <= min(GRIDY-1, d_gridInd[p*3+1]+1); ++j){
            for(int k = max(0, d_gridInd[p*3+2]-1); k <= min(GRIDZ-1, d_gridInd[p*3+2]+1); ++k){
                gi = get1DGridIdx(i, j, k);
                for(int n = 0; n < d_gridCount[gi]; ++n){ // for all particles in the grid
                    gradL = glm::length(calcGradConstraint(pi, d_ppos[d_grid[gi * MAX_NEIGHBORS + n]]));
                    gradSum += gradL * gradL;
                }
            }
        }
    }
     
    gradL = glm::length(calcGradConstraintAtI(p, d_ppos, d_grid, d_gridCount, d_gridInd));
    gradSum += gradL * gradL;
    d_lambda[p] = top / (gradSum + EPS);
} 
 
__global__ void calcdPos(Vector3s *d_ppos, Vector3s *d_dpos, int *d_grid, int *d_gridCount, int *d_gridInd, scalar *d_lambda){
    int p = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(p >= NUM_PARTICLES)
        return;

    Vector3s dp(0.0, 0.0, 0.0);
    int q = 0; 
    int gi = 0;
    scalar plambda = d_lambda[p]; 
    Vector3s pi = d_ppos[p];
    Vector3s pj; 

    scalar scorr = 0; // bla 

    for(int i = max(0, d_gridInd[p*3]-1); i <= min(GRIDX-1, d_gridInd[p*3]+1); ++i){
        for(int j = max(0, d_gridInd[p*3+1]-1); j <= min(GRIDY-1, d_gridInd[p*3+1]+1); ++j){
            for(int k = max(0, d_gridInd[p*3+2]-1); k <= min(GRIDZ-1, d_gridInd[p*3+2]+1); ++k){
                gi = get1DGridIdx(i, j, k);
                for(int n = 0; n < d_gridCount[gi]; ++n){ // for all particles in the grid
                    q = d_grid[gi * MAX_NEIGHBORS + n];
                    pj = d_ppos[q];                
    
                #if ART_PRESSURE > 0
                    scalar top = wPoly6Kernel(pi, pj); 
                    scorr = - K * (pow(top / QSCALE, N)); 
                #endif

                    dp += (plambda + d_lambda[q] + scorr) * wSpikyKernelGrad(pi, pj);
                }
            }
        }
    }
    d_dpos[p] = dp / (scalar) P0;
    //d_dpos[p] = Vector3s(.1, 0, 0);
}

__global__ void updateForReals(Vector3s* d_pos, Vector3s* d_vel, Vector3s* d_ppos, scalar dt){
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(id < NUM_PARTICLES){
        d_vel[id] = (d_ppos[id] - d_pos[id])/dt;
        d_pos[id] = d_ppos[id];
    }
}

__global__ void updateXSPHAndOmega(Vector3s *d_pos, Vector3s *d_vel, Vector3s *d_omega, int *d_grid, int *d_gridCount, int *d_gridInd){
    int p = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(p >= NUM_PARTICLES)
        return;
    
    Vector3s dv(0.0, 0.0, 0.0);
    Vector3s vi = d_vel[p];
    Vector3s pi = d_pos[p];
    Vector3s pj; 
    int gi; 
    int q; 
    Vector3s vij;

    #if VORTICITY > 0
    Vector3s omega(0.0, 0.0, 0.0); 
    #endif
    

    for(int i = max(0, d_gridInd[p*3]-1); i <= min(GRIDX-1, d_gridInd[p*3]+1); ++i){
        for(int j = max(0, d_gridInd[p*3+1]-1); j <= min(GRIDY-1, d_gridInd[p*3+1]+1); ++j){
            for(int k = max(0, d_gridInd[p*3+2]-1); k <= min(GRIDZ-1, d_gridInd[p*3+2]+1); ++k){
                gi = get1DGridIdx(i, j, k);
                for(int n = 0; n < d_gridCount[gi]; ++n){ // for all particles in the grid
                    q = d_grid[gi * MAX_NEIGHBORS + n];
                    vij = vi - d_vel[q];
                    pj = d_pos[q]; 
                    dv += vij * wPoly6Kernel(pi, pj);    

                    #if VORTICITY > 0
                    omega += glm::cross(vij, wSpikyKernelGrad(pi, pj)); 
                    #endif          
                }
            }
        }
    }

    dv *= C;
    d_vel[p] += dv; 
    #if VORTICITY > 0
    d_omega[p] = omega;
    #endif
}


__global__ void applyVorticity(Vector3s *d_pos, Vector3s *d_vel, Vector3s *d_omega, int *d_grid, int *d_gridCount, int *d_gridInd, scalar dt){
    int p = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(p >= NUM_PARTICLES)
        return;

    /*
    Vector3s pi = d_pos[p];
    Vector3s pj; 
    Vector3s omega = d_omega[p]; 
    scalar dom;
    int gi;
    int q;
    Vector3s vort(0, 0, 0); 
    Vector3s dp;
    

    for(int i = max(0, d_gridInd[p*3]-1); i <= min(GRIDX-1, d_gridInd[p*3]+1); ++i){
        for(int j = max(0, d_gridInd[p*3+1]-1); j <= min(GRIDY-1, d_gridInd[p*3+1]+1); ++j){
            for(int k = max(0, d_gridInd[p*3+2]-1); k <= min(GRIDZ-1, d_gridInd[p*3+2]+1); ++k){
                gi = get1DGridIdx(i, j, k);
                for(int n = 0; n < d_gridCount[gi]; ++n){ // for all particles in the grid
                    q = d_grid[gi * MAX_NEIGHBORS + n];
                    pj = d_pos[q]; 
                    dp = pj - pi; 
                    dom = glm::length(omega - d_omega[q]);     
                    vort[0] += dom / dp[0];
                    vort[1] += dom / dp[1];
                    vort[2] += dom / dp[2];                      
                }
            }
        }
    }
    vort /= (glm::length(vort) + EPS);     
    //d_vel[p] += (scalar)(dt * VORT_EPS / FP_MASS) * (glm::cross(vort, omega)); 
*/
}


void initGPUFluid(){
    // allocate memory on GPU
    // Initialize positions, velocities
    std::cout << "GPU: Initializing things" << std::endl;

    GPU_CHECKERROR(cudaMalloc((void **)&d_pos, NUM_PARTICLES * sizeof(Vector3s)));
    GPU_CHECKERROR(cudaMalloc((void **)&d_vel, NUM_PARTICLES * sizeof(Vector3s)));
    GPU_CHECKERROR(cudaMalloc((void **)&d_ppos, NUM_PARTICLES * sizeof(Vector3s)));
    GPU_CHECKERROR(cudaMalloc((void **)&d_dpos, NUM_PARTICLES * sizeof(Vector3s)));
    GPU_CHECKERROR(cudaMalloc((void **)&d_pcalc, NUM_PARTICLES * sizeof(scalar)));
    GPU_CHECKERROR(cudaMalloc((void **)&d_lambda, NUM_PARTICLES * sizeof(scalar)));
    #if VORTICITY > 0 
    GPU_CHECKERROR(cudaMalloc((void **)&d_omega, NUM_PARTICLES * sizeof(Vector3s))); 
    #endif


    grid_X = ceil(WIDTH/H);
    grid_Y = ceil(HEIGHT/H);
    grid_Z = ceil(DEPTH/H); 
    GPU_CHECKERROR(cudaMemcpyToSymbol(GRIDX, &grid_X, sizeof(int)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(GRIDY, &grid_Y, sizeof(int)));
    GPU_CHECKERROR(cudaMemcpyToSymbol(GRIDZ, &grid_Z, sizeof(int)));


    GPU_CHECKERROR(cudaMalloc((void **)&d_grid, grid_X * grid_Y * grid_Z * MAX_NEIGHBORS * sizeof(int)));
    GPU_CHECKERROR(cudaMalloc((void **)&d_gridCount, grid_X * grid_Y * grid_Z *sizeof(int)));
    GPU_CHECKERROR(cudaMalloc((void **)&d_gridInd, 3 * NUM_PARTICLES * sizeof(int)));

    GPU_CHECKERROR(cudaMemset((void *)d_vel, 0, NUM_PARTICLES * sizeof(Vector3s)));

    Vector3s dq(H, 0, 0);
    dq *= (scalar)DQ;
    scalar q_scale = wPoly6Kernel(Vector3s(0, 0, 0), dq);
    GPU_CHECKERROR(cudaMemcpyToSymbol(QSCALE, &q_scale, sizeof(scalar)));


    /*
    curandState *state;
    curandGenerator_t gen;
    curandCreateGenerator(&gen, CURAND_RNG_PSEUDO_DEFAULT);
    curandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    curandGenerateUniform(gen, d_pcalc, NUM_PARTICLES); // temporarily store here I guess
    curandDestroyGenerator(gen);
    */

    Vector3s *h_pos;
    GPU_CHECKERROR(cudaMallocHost((void **)&h_pos, NUM_PARTICLES * sizeof(Vector3s)));
    float x; 
    float y; 
    float z;
    for(int i = 0; i < NUM_PARTICLES; ++i){
         x = static_cast <float> (rand()) / static_cast<float>(RAND_MAX/9.0);
         y = static_cast <float> (rand()) / static_cast<float>(RAND_MAX/9.0);
         z = static_cast <float> (rand()) / static_cast<float>(RAND_MAX/9.0);
         h_pos[i] = Vector3s(x, y, z);
    }
    GPU_CHECKERROR(cudaMemcpy((void *)d_pos, (void *)h_pos, NUM_PARTICLES * sizeof(Vector3s), cudaMemcpyHostToDevice));

    GPU_CHECKERROR(cudaFreeHost(h_pos));


}

void preserveOwnBoundary(bool update){
    int gridSize = ceil((NUM_PARTICLES * 1.0)/(BLOCKSIZE*1.0));
    if(update)
        preserveFluidBoundaryWithUpdate<<<gridSize, BLOCKSIZE>>>(d_pos, d_ppos, d_dpos);    
    else
        preserveFluidBoundary<<<gridSize, BLOCKSIZE>>>(d_pos, d_ppos, d_dpos);

    GPU_CHECKERROR(cudaGetLastError());
    GPU_CHECKERROR(cudaThreadSynchronize());



} 



void updatePredFromForce(scalar dt){
    int gridSize = ceil((NUM_PARTICLES * 1.0)/(BLOCKSIZE*1.0));
    updateFromForce<<<gridSize, BLOCKSIZE>>>(d_pos, d_vel, d_ppos, dt, Vector3s(0.f, FP_MASS*-10.0f, 0.f));    
    //updateFromForce<<<gridSize, BLOCKSIZE>>>(d_pos, d_vel, d_ppos, dt, Vector3s(0.f, 0.0f, 0.f));    
    GPU_CHECKERROR(cudaGetLastError());
    GPU_CHECKERROR(cudaThreadSynchronize());


}

void updateValForReals(scalar dt){
    int gridSize = ceil((NUM_PARTICLES * 1.0)/(BLOCKSIZE*1.0));
    updateForReals<<<gridSize, BLOCKSIZE>>>(d_pos, d_vel, d_ppos, dt);    
    GPU_CHECKERROR(cudaGetLastError());
    GPU_CHECKERROR(cudaThreadSynchronize());


}

void buildGrid(){
    GPU_CHECKERROR(cudaMemset((void *)d_grid, -1, grid_X*grid_Y*grid_Z*MAX_NEIGHBORS*sizeof(int)));
    GPU_CHECKERROR(cudaMemset((void *)d_gridCount, 0, grid_X * grid_Y * grid_Z *sizeof(int)));

    int gridSize = ceil((NUM_PARTICLES * 1.0)/(BLOCKSIZE*1.0));
    buildGrid<<<gridSize, BLOCKSIZE>>>(d_ppos, d_grid, d_gridCount, d_gridInd);
    GPU_CHECKERROR(cudaGetLastError());
    GPU_CHECKERROR(cudaThreadSynchronize());

     
    

}



void calculatePressures(){
    int gridSize = ceil((NUM_PARTICLES * 1.0)/(BLOCKSIZE*1.0)); 
    calcPressures<<<gridSize, BLOCKSIZE>>>(d_ppos, d_grid, d_gridCount, d_gridInd, d_pcalc);  
    GPU_CHECKERROR(cudaGetLastError());
    GPU_CHECKERROR(cudaThreadSynchronize());

       
}

void calculateLambdas(){
    GPU_CHECKERROR(cudaMemset((void *)d_lambda, 0, NUM_PARTICLES * sizeof(scalar)));
    int gridSize = ceil((NUM_PARTICLES * 1.0)/(BLOCKSIZE*1.0)); 
    calcLambdas<<<gridSize, BLOCKSIZE>>>(d_ppos, d_grid, d_gridCount, d_gridInd, d_pcalc, d_lambda); 
    GPU_CHECKERROR(cudaGetLastError());
    GPU_CHECKERROR(cudaThreadSynchronize());



}

void calculatedPos(){
    GPU_CHECKERROR(cudaMemset((void *)d_dpos, 0, NUM_PARTICLES * sizeof(Vector3s)));
    int gridSize = ceil((NUM_PARTICLES * 1.0)/(BLOCKSIZE*1.0)); 
    calcdPos<<<gridSize, BLOCKSIZE>>>(d_ppos, d_dpos, d_grid, d_gridCount, d_gridInd, d_lambda); 
    GPU_CHECKERROR(cudaGetLastError());
    GPU_CHECKERROR(cudaThreadSynchronize());
}

void applydPToPredPos(){
    int gridSize = ceil((NUM_PARTICLES * 1.0)/(BLOCKSIZE*1.0)); 
    applydPToPPos<<<gridSize, BLOCKSIZE>>>(d_ppos, d_dpos); 
    GPU_CHECKERROR(cudaGetLastError());
    GPU_CHECKERROR(cudaThreadSynchronize());
}

void adjustVel(scalar dt){
    #if XSPH == 0
    return;
    #endif

    int gridSize = ceil((NUM_PARTICLES * 1.0)/(BLOCKSIZE*1.0)); 
    updateXSPHAndOmega<<<gridSize, BLOCKSIZE>>>(d_pos, d_vel, d_omega, d_grid, d_gridCount, d_gridInd); 
    GPU_CHECKERROR(cudaGetLastError());
    GPU_CHECKERROR(cudaThreadSynchronize());
    
    #if VORTICITY == 0
    return;
    #endif
 
    applyVorticity<<<gridSize, BLOCKSIZE>>>(d_pos, d_vel, d_omega, d_grid, d_gridCount, d_gridInd, dt); 
    GPU_CHECKERROR(cudaGetLastError());
    GPU_CHECKERROR(cudaThreadSynchronize());


}

void stepSystemGPUFluid(scalar dt){
    if(!deviceHappy)
        return;

    updatePredFromForce(dt);    

    GPU_CHECKERROR(cudaMemset((void *)d_dpos, 0, NUM_PARTICLES * sizeof(Vector3s)));
    preserveOwnBoundary(true); 
    
    
    buildGrid(); 
    
    for(int loop = 0; loop < 1; ++loop){
        calculatePressures();
        calculateLambdas();
        calculatedPos();
        preserveOwnBoundary(false);
        applydPToPredPos();
    }
    
    updateValForReals(dt); 
    adjustVel(dt);
}



void updateVBOGPUFluid(float *vboptr){
    int gridSize = ceil((NUM_PARTICLES * 1.0)/(BLOCKSIZE*1.0)); 
    sendToVBO<<<gridSize, BLOCKSIZE>>>(vboptr, d_pos);  

    cudaError_t err = cudaGetLastError();
    if(err != cudaSuccess){
        deviceHappy = false;
        fprintf(stderr, "%s in %s at line %d\n", cudaGetErrorString(err), __FILE__, __LINE__);
        std::cout << "vboptr: " << vboptr << std::endl;
        return;
    }
    else{
        deviceHappy = true;
    }
    // Is sad the first call, then fine
    GPU_CHECKERROR(cudaThreadSynchronize());

}

void cleanUpGPUFluid(){

    GPU_CHECKERROR(cudaFree(d_pos));
    GPU_CHECKERROR(cudaFree(d_vel));
    GPU_CHECKERROR(cudaFree(d_ppos));
    GPU_CHECKERROR(cudaFree(d_dpos));
    GPU_CHECKERROR(cudaFree(d_pcalc));
    GPU_CHECKERROR(cudaFree(d_lambda));

    GPU_CHECKERROR(cudaFree(d_grid));
    GPU_CHECKERROR(cudaFree(d_gridCount));
    GPU_CHECKERROR(cudaFree(d_gridInd));
        
    #if VORTICITY > 0
    GPU_CHECKERROR(cudaFree(d_omega));
    #endif

}

#endif
